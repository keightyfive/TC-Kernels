#include "hip/hip_runtime.h"


/*
 *  Global Switches
 */

#define SAC_DO_CHECK           0
#define SAC_DO_CHECK_TYPE      0
#define SAC_DO_CHECK_GPU       0
#define SAC_DO_CHECK_BOUNDARY   0
#define SAC_DO_CHECK_MALLOC     0
#define SAC_DO_CHECK_ERRNO      0
#define SAC_DO_CHECK_HEAP       0
#define SAC_DO_CHECK_DISTMEM    0
#define SAC_DO_CHECK_DISTMEMPHM 0

#define SAC_DO_PHM             0
#define SAC_DO_APS             0
#define SAC_DO_DAO             0
#define SAC_DO_MSCA            0

#define SAC_DO_PROFILE         0
#define SAC_DO_PROFILE_WITH    0
#define SAC_DO_PROFILE_FUN     0
#define SAC_DO_PROFILE_INL     0
#define SAC_DO_PROFILE_LIB     0
#define SAC_DO_PROFILE_DISTMEM 0

#define SAC_DO_TRACE           0
#define SAC_DO_TRACE_REF       0
#define SAC_DO_TRACE_MEM       0
#define SAC_DO_TRACE_PRF       0
#define SAC_DO_TRACE_FUN       0
#define SAC_DO_TRACE_WL        0
#define SAC_DO_TRACE_AA        0
#define SAC_DO_TRACE_MT        0
#define SAC_DO_TRACE_RTSPEC    0
#define SAC_DO_TRACE_DISTMEM   0
#define SAC_DO_CACHESIM        0
#define SAC_DO_CACHESIM_ADV    0
#define SAC_DO_CACHESIM_GLOBAL 1
#define SAC_DO_CACHESIM_FILE   0
#define SAC_DO_CACHESIM_PIPE   0
#define SAC_DO_CACHESIM_IMDT   1

#define SAC_DO_MULTITHREAD     0
#define SAC_DO_MT_PTHREAD      0
#define SAC_DO_MT_LPEL         0
#define SAC_DO_MT_OMP          0
#define SAC_DO_DISTMEM         0
#define SAC_DO_DISTMEM_GASNET  0
#define SAC_DO_DISTMEM_GPI     0
#define SAC_DO_DISTMEM_MPI     0
#define SAC_DO_DISTMEM_ARMCI   0
#define SAC_DO_DISTMEM_ALLOC_CACHE_OUTSIDE_DSM 0
#define SAC_DO_DISTMEM_PTR_DESC 0
#define SAC_DO_DISTMEM_PTR_CACHE 0
#define SAC_DO_THREADS_STATIC  1
#define SAC_DO_FP              0
#define SAC_DO_MT_CREATE_JOIN  0
#define SAC_DEBUG_RC           0


/*
 *  Global Settings
 */

#define SAC_FORCE_DESC_SIZE -1


/*
 *  MUTC Backend Specific Switches
 */

#define SAC_MUTC_FUNAP_AS_CREATE  0
#define SAC_MUTC_THREAD_MALLOC 0
#define SAC_MUTC_DISABLE_THREAD_MEM 0
#define SAC_MUTC_BENCH 0
#define SAC_MUTC_MACROS  0
#define SAC_MUTC_RC_PLACES  1
#define SAC_MUTC_RC_INDIRECT  0
#define SAC_MUTC_SEQ_DATA_PARALLEL  0
#define SAC_MUTC_FORCE_SPAWN_FLAGS

#define SAC_CUDA_MACROS  1
#define SAC_OMP_MACROS  0

#define SAC_DO_COMPILE_MODULE  0
#define SAC_C_EXTERN           extern "C"



/*
 *  Global Settings
 */

#ifndef NULL
#  ifdef __cplusplus
#    define NULL         0
#  else
#    define NULL         (void*) 0
#  endif
#endif

#define SAC_SET_TMPDIR              "/tmp"
#define SAC_SET_INITIAL_MASTER_HEAPSIZE      1048576
#define SAC_SET_INITIAL_WORKER_HEAPSIZE      65536
#define SAC_SET_INITIAL_UNIFIED_HEAPSIZE     0

#ifndef SAC_SET_RTSPEC_THREADS
#define SAC_SET_RTSPEC_THREADS              1
#endif

#ifndef SAC_SET_MTMODE
#define SAC_SET_MTMODE               0
#endif

#define SAC_SET_CPU_BIND_STRATEGY 0
#define SAC_SET_BARRIER_TYPE               0
#define SAC_SET_SMART_DECISIONS            0
#define SAC_SET_SMART_FILENAME           "default"
#define SAC_SET_SMART_ARCH               "(null)"
#define SAC_SET_SMART_PERIOD               500
#ifndef SAC_SET_THREADS_MAX
#define SAC_SET_THREADS_MAX          128
#endif

#ifndef SAC_SET_THREADS
#define SAC_SET_THREADS              1
#endif

#ifndef SAC_OMP_ACTIVE_LEVEL
#define SAC_OMP_ACTIVE_LEVEL          1
#endif

#ifndef SAC_SET_MASTERCLASS
#define SAC_SET_MASTERCLASS          0
#endif

#define SAC_SET_NUM_SCHEDULERS       0

#define SAC_SET_CACHE_1_SIZE         -1
#define SAC_SET_CACHE_1_LINE         4
#define SAC_SET_CACHE_1_ASSOC        1
#define SAC_SET_CACHE_1_WRITEPOL     SAC_CS_default
#define SAC_SET_CACHE_1_MSCA_FACTOR  0.00

#define SAC_SET_CACHE_2_SIZE         -1
#define SAC_SET_CACHE_2_LINE         4
#define SAC_SET_CACHE_2_ASSOC        1
#define SAC_SET_CACHE_2_WRITEPOL     SAC_CS_default
#define SAC_SET_CACHE_2_MSCA_FACTOR  0.00

#define SAC_SET_CACHE_3_SIZE         -1
#define SAC_SET_CACHE_3_LINE         4
#define SAC_SET_CACHE_3_ASSOC        1
#define SAC_SET_CACHE_3_WRITEPOL     SAC_CS_default
#define SAC_SET_CACHE_3_MSCA_FACTOR  0.00

#define SAC_SET_CACHESIM_HOST        ""
#define SAC_SET_CACHESIM_FILE        "tbmm.cs"
#define SAC_SET_CACHESIM_DIR         "/tmp"
#define SAC_SET_MAXFUN               0
#define SAC_SET_MAXFUNAP             1
#define SBLOCKSZ               16
#define LBLOCKSZ               256



/*
 *  Includes
 */


#include "sac.h"


#if SAC_OMP_MACROS

#include "omp.h"

#endif

#if SAC_CUDA_MACROS

#include <stdio.h>


#include <hip/hip_runtime.h>


#include <hip/hip_runtime.h>

#endif

/*
 *  SAC-Program tbmm.sac :
 */


/* Additional headers for external function declarations */
#include <stdlib.h>


/*
 *  type definitions
 */

SAC_ND_TYPEDEF((SACt_sacprelude_p__SACarg, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SAC_hidden)
SAC_ND_TYPEDEF((SACt_Random__Random, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SAC_hidden)
SAC_ND_TYPEDEF((SACt_ComplexBasics__complex, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (DOU, )))))))))), double)
SAC_ND_TYPEDEF((SACt_Complex__complex, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (DOU, )))))))))), double)
SAC_ND_TYPEDEF((SACt_World__World, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SAC_hidden)
SAC_ND_TYPEDEF((SACt_C99Benchmarking__C99Benchmarking, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SAC_hidden)
SAC_ND_TYPEDEF((SACt_String__string, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SAC_hidden)
SAC_ND_TYPEDEF((SACt_Interval__Interval, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SAC_hidden)
SAC_ND_TYPEDEF((SACt_C99Benchmarking__Interval, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_Interval__Interval)
SAC_ND_TYPEDEF((SACt_MTClock__MTClock, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SAC_hidden)
SAC_ND_TYPEDEF((SACt_Terminal__Terminal, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SAC_hidden)
SAC_ND_TYPEDEF((SACt_TermFile__TermFile, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SAC_hidden)


/*
 *  Global Definitions
 */

SAC_MT_DEFINE()
SAC_PF_DEFINE()
SAC_HM_DEFINE()


/*
 *  prototypes for externals (FUNDECS)
 */

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACrandom, (int, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 2, in_nodesc, int, (SACl_MIN, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in_nodesc, int, (SACl_MAX, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACrandom, SAC_ND_TYPE_NT( (int, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))),  SAC_ND_PARAM_in_nodesc( (SACl_MIN, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in_nodesc( (SACl_MAX, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( to_string, , 3, out, SACt_String__string, (SAC_arg_1, (SCL, (HID, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), in, unsigned char, (SACl_A, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in_nodesc, int, (SACl_LENGTH, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( to_string, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (HID, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), SACt_String__string), SAC_ND_PARAM_in( (SACl_A, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in_nodesc( (SACl_LENGTH, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));



/*
 *  prototypes for locals (FUNDEFS)
 */

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__partitionSlicer__i_S__i_S__i_S__i_S__i_S, , 9, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_3, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_4, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_min, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_max, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_axis, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_lb, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ub, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__partitionSlicer__i_S__i_S__i_S__i_S__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out( (SAC_arg_2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out( (SAC_arg_3, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out( (SAC_arg_4, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_min, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_max, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_axis, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_lb, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_ub, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__partitionIntersectMax__i_S__i_S, , 3, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_PWLbound1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ivmin, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__partitionIntersectMax__i_S__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_PWLbound1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_ivmin, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__isPartitionIntersectNull__i_S__i_S__i_S__i_S, , 5, out, bool, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idxmin, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idxmax, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound2, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__isPartitionIntersectNull__i_S__i_S__i_S__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_idxmin, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_idxmax, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_bound1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_bound2, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selSxADistmemLocal__i_S__i_S, , 3, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selSxADistmemLocal__i_S__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selSxADistmemLocal__i_S__f_S, , 3, out, float, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, float, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selSxADistmemLocal__i_S__f_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selSxADistmemLocal__i_S__d_S, , 3, out, double, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, double, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selSxADistmemLocal__i_S__d_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selSxADistmemLocal__i_S__bl_S, , 3, out, bool, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selSxADistmemLocal__i_S__bl_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selSxADistmemLocal__i_S__c_S, , 3, out, unsigned char, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, unsigned char, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selSxADistmemLocal__i_S__c_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selSxADistmemLocal__i_S__b_S, , 3, out, byte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, byte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selSxADistmemLocal__i_S__b_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selSxADistmemLocal__i_S__s_S, , 3, out, short, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, short, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selSxADistmemLocal__i_S__s_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selSxADistmemLocal__i_S__l_S, , 3, out, long, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, long, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selSxADistmemLocal__i_S__l_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selSxADistmemLocal__i_S__ll_S, , 3, out, longlong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, longlong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selSxADistmemLocal__i_S__ll_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selSxADistmemLocal__i_S__ub_S, , 3, out, ubyte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ubyte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selSxADistmemLocal__i_S__ub_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selSxADistmemLocal__i_S__us_S, , 3, out, ushort, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ushort, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selSxADistmemLocal__i_S__us_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selSxADistmemLocal__i_S__ui_S, , 3, out, uint, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, uint, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selSxADistmemLocal__i_S__ui_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selSxADistmemLocal__i_S__ul_S, , 3, out, ulong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selSxADistmemLocal__i_S__ul_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selSxADistmemLocal__i_S__ull_S, , 3, out, ulonglong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulonglong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selSxADistmemLocal__i_S__ull_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__partitionIntersectMin__i_S__i_S, , 3, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_PWLbound2, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ivmax, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__partitionIntersectMin__i_S__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_PWLbound2, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_ivmax, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__adjustLacFunParams__bl_S__i_S__i_S, , 4, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_p, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_i, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_iv, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__adjustLacFunParams__bl_S__i_S__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_p, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_i, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_iv, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf__MAIN__main, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__prod__i_S, , 2, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_v, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__prod__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_v, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__adjustLacFunParamsReshape__bl_S__i_S__i_S__i_S, , 5, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_p, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_i, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_iv, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_shp, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__adjustLacFunParamsReshape__bl_S__i_S__i_S__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_p, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_i, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_iv, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_shp, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__eq__i_S__i_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__eq__i_S__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__eq__f_S__f_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, float, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_B, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__eq__f_S__f_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__eq__d_S__d_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, double, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, double, (SACl_B, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__eq__d_S__d_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__eq__bl_S__bl_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__eq__bl_S__bl_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__eq__c_S__c_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, unsigned char, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, unsigned char, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__eq__c_S__c_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__eq__b_S__b_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, byte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, byte, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__eq__b_S__b_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__eq__s_S__s_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, short, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, short, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__eq__s_S__s_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__eq__l_S__l_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, long, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, long, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__eq__l_S__l_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__eq__ll_S__ll_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, longlong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, longlong, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__eq__ll_S__ll_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__eq__ub_S__ub_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, ubyte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, ubyte, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__eq__ub_S__ub_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__eq__us_S__us_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, ushort, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, ushort, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__eq__us_S__us_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__eq__ui_S__ui_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, uint, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, uint, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__eq__ui_S__ui_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__eq__ul_S__ul_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, ulong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, ulong, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__eq__ul_S__ul_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__eq__ull_S__ull_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, ulonglong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, ulonglong, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__eq__ull_S__ull_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__gridFiller__i_S__i_S__i_S__i_S__i_S, , 8, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_3, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_lb, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ub, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_wdth, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_dim, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_maxwidth, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__gridFiller__i_S__i_S__i_S__i_S__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out( (SAC_arg_2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out( (SAC_arg_3, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_lb, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_ub, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_wdth, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_dim, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_maxwidth, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__isPartitionIntersect1Part__i_S__i_S__i_S__i_S, , 5, out, bool, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idxmin, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idxmax, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound2, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__isPartitionIntersect1Part__i_S__i_S__i_S__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_idxmin, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_idxmax, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_bound1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_bound2, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf__MAIN_CLsacprelude_p__zero__i_S, , 2, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf__MAIN_CLsacprelude_p__zero__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf__MAIN_CLsacprelude_p__zero__d_S, , 2, out, double, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, double, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf__MAIN_CLsacprelude_p__zero__d_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf__MAIN_CLsacprelude_p__zero__f_S, , 2, out, float, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf__MAIN_CLsacprelude_p__zero__f_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf__MAIN_CLsacprelude_p__zero__c_S, , 2, out, unsigned char, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, unsigned char, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf__MAIN_CLsacprelude_p__zero__c_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf__MAIN_CLsacprelude_p__zero__bl_S, , 2, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf__MAIN_CLsacprelude_p__zero__bl_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf__MAIN_CLsacprelude_p__zero__b_S, , 2, out, byte, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, byte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf__MAIN_CLsacprelude_p__zero__b_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf__MAIN_CLsacprelude_p__zero__s_S, , 2, out, short, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, short, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf__MAIN_CLsacprelude_p__zero__s_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf__MAIN_CLsacprelude_p__zero__l_S, , 2, out, long, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, long, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf__MAIN_CLsacprelude_p__zero__l_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf__MAIN_CLsacprelude_p__zero__ll_S, , 2, out, longlong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, longlong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf__MAIN_CLsacprelude_p__zero__ll_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf__MAIN_CLsacprelude_p__zero__ub_S, , 2, out, ubyte, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, ubyte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf__MAIN_CLsacprelude_p__zero__ub_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf__MAIN_CLsacprelude_p__zero__us_S, , 2, out, ushort, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, ushort, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf__MAIN_CLsacprelude_p__zero__us_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf__MAIN_CLsacprelude_p__zero__ui_S, , 2, out, uint, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, uint, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf__MAIN_CLsacprelude_p__zero__ui_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf__MAIN_CLsacprelude_p__zero__ul_S, , 2, out, ulong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, ulong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf__MAIN_CLsacprelude_p__zero__ul_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf__MAIN_CLsacprelude_p__zero__ull_S, , 2, out, ulonglong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, ulonglong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf__MAIN_CLsacprelude_p__zero__ull_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__partitionMax__i_S__i_S, , 3, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_x, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_y, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__partitionMax__i_S__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_x, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_y, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__partitionMin__i_S__i_S, , 3, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_x, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_y, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__partitionMin__i_S__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_x, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_y, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___PL_PL__i_S, , 2, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_a, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___PL_PL__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_a, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__sel__i_S__i_S, , 3, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__sel__i_S__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__sel__i_S__f_S, , 3, out, float, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, float, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__sel__i_S__f_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__sel__i_S__d_S, , 3, out, double, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, double, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__sel__i_S__d_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__sel__i_S__bl_S, , 3, out, bool, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__sel__i_S__bl_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__sel__i_S__c_S, , 3, out, unsigned char, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, unsigned char, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__sel__i_S__c_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__sel__i_S__b_S, , 3, out, byte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, byte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__sel__i_S__b_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__sel__i_S__s_S, , 3, out, short, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, short, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__sel__i_S__s_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__sel__i_S__l_S, , 3, out, long, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, long, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__sel__i_S__l_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__sel__i_S__ll_S, , 3, out, longlong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, longlong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__sel__i_S__ll_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__sel__i_S__ub_S, , 3, out, ubyte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ubyte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__sel__i_S__ub_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__sel__i_S__us_S, , 3, out, ushort, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ushort, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__sel__i_S__us_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__sel__i_S__ui_S, , 3, out, uint, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, uint, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__sel__i_S__ui_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__sel__i_S__ul_S, , 3, out, ulong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__sel__i_S__ul_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p__sel__i_S__ull_S, , 3, out, ulonglong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulonglong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p__sel__i_S__ull_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selVxADistmemLocal__i_S__i_S, , 3, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selVxADistmemLocal__i_S__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selVxADistmemLocal__i_S__f_S, , 3, out, float, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, float, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selVxADistmemLocal__i_S__f_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selVxADistmemLocal__i_S__d_S, , 3, out, double, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, double, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selVxADistmemLocal__i_S__d_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selVxADistmemLocal__i_S__bl_S, , 3, out, bool, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selVxADistmemLocal__i_S__bl_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selVxADistmemLocal__i_S__c_S, , 3, out, unsigned char, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, unsigned char, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selVxADistmemLocal__i_S__c_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selVxADistmemLocal__i_S__b_S, , 3, out, byte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, byte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selVxADistmemLocal__i_S__b_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selVxADistmemLocal__i_S__s_S, , 3, out, short, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, short, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selVxADistmemLocal__i_S__s_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selVxADistmemLocal__i_S__l_S, , 3, out, long, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, long, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selVxADistmemLocal__i_S__l_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selVxADistmemLocal__i_S__ll_S, , 3, out, longlong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, longlong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selVxADistmemLocal__i_S__ll_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selVxADistmemLocal__i_S__ub_S, , 3, out, ubyte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ubyte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selVxADistmemLocal__i_S__ub_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selVxADistmemLocal__i_S__us_S, , 3, out, ushort, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ushort, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selVxADistmemLocal__i_S__us_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selVxADistmemLocal__i_S__ui_S, , 3, out, uint, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, uint, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selVxADistmemLocal__i_S__ui_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selVxADistmemLocal__i_S__ul_S, , 3, out, ulong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selVxADistmemLocal__i_S__ul_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACwf_sacprelude_p___selVxADistmemLocal__i_S__ull_S, , 3, out, ulonglong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, int, (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulonglong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2( SACwf_sacprelude_p___selVxADistmemLocal__i_S__ull_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in( (SACl_idx, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf__MAIN__tbmm__f_X_X_X__f_X_X_X, , 3, out, float, (SAC_arg_1, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf__MAIN__tbmm__f_X_X_X__f_X_X_X, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf__MAIN__main, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf__MAIN_CL_INIT__init, , 0)
 */
SAC_ND_DECL_FUN2( SACf__MAIN_CL_INIT__init, void, void);

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__zero__ull_S, , 2, out, ulonglong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, ulonglong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__zero__ull_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__zero__ul_S, , 2, out, ulong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, ulong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__zero__ul_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__zero__ui_S, , 2, out, uint, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, uint, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__zero__ui_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__zero__us_S, , 2, out, ushort, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, ushort, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__zero__us_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__zero__ub_S, , 2, out, ubyte, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, ubyte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__zero__ub_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__zero__ll_S, , 2, out, longlong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, longlong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__zero__ll_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__zero__l_S, , 2, out, long, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, long, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__zero__l_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__zero__s_S, , 2, out, short, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, short, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__zero__s_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__zero__b_S, , 2, out, byte, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, byte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__zero__b_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__zero__bl_S, , 2, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__zero__bl_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__zero__c_S, , 2, out, unsigned char, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, unsigned char, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__zero__c_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__zero__f_S, , 2, out, float, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__zero__f_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__zero__d_S, , 2, out, double, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, double, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__zero__d_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__zero__i_S, , 2, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__zero__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_World_CL_INIT__init_TheWorld__SACt_World__World, , 1, inout, SACt_World__World, (SACp_OI_object, (SCL, (HID, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_World_CL_INIT__init_TheWorld__SACt_World__World, void,  SAC_ND_PARAM_inout( (SACp_OI_object, (SCL, (HID, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), SACt_World__World));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_C99Benchmarking_CL_INIT__init_TheBenchmarkObject__SACt_C99Benchmarking__C99Benchmarking, , 1, inout, SACt_C99Benchmarking__C99Benchmarking, (SACp_OI_object, (SCL, (HID, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_C99Benchmarking_CL_INIT__init_TheBenchmarkObject__SACt_C99Benchmarking__C99Benchmarking, void,  SAC_ND_PARAM_inout( (SACp_OI_object, (SCL, (HID, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), SACt_C99Benchmarking__C99Benchmarking));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_C99Benchmarking__getInterval__SACt_String__string__i__i, , 4, out, SACt_Interval__Interval, (SAC_arg_1, (SCL, (HID, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), in, SACt_String__string, (SACl_interval_name, (SCL, (HID, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), in, int, (SACl_interval_number, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_unit_time, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_C99Benchmarking__getInterval__SACt_String__string__i__i, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (HID, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), SACt_Interval__Interval), SAC_ND_PARAM_in( (SACl_interval_name, (SCL, (HID, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (OTH, )))))))))), SACt_String__string), SAC_ND_PARAM_in( (SACl_interval_number, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_unit_time, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_MTClock_CL_INIT__init_TheMTClock__SACt_MTClock__MTClock, , 1, inout, SACt_MTClock__MTClock, (SACp_OI_object, (SCL, (HID, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_MTClock_CL_INIT__init_TheMTClock__SACt_MTClock__MTClock, void,  SAC_ND_PARAM_inout( (SACp_OI_object, (SCL, (HID, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), SACt_MTClock__MTClock));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_C99Benchmarking__start__SACt_C99Benchmarking__Interval, , 1, inout, SACt_Interval__Interval, (SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_C99Benchmarking__start__SACt_C99Benchmarking__Interval, void,  SAC_ND_PARAM_inout( (SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), SACt_Interval__Interval));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_C99Benchmarking__end__SACt_C99Benchmarking__Interval, , 1, inout, SACt_Interval__Interval, (SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_C99Benchmarking__end__SACt_C99Benchmarking__Interval, void,  SAC_ND_PARAM_inout( (SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), SACt_Interval__Interval));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_Terminal_CL_INIT__init_TheTerminal__SACt_Terminal__Terminal, , 1, inout, SACt_Terminal__Terminal, (SACp_OI_object, (SCL, (HID, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_Terminal_CL_INIT__init_TheTerminal__SACt_Terminal__Terminal, void,  SAC_ND_PARAM_inout( (SACp_OI_object, (SCL, (HID, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), SACt_Terminal__Terminal));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_TermFile_CL_INIT__init_stdout__SACt_TermFile__TermFile, , 1, inout, SACt_TermFile__TermFile, (SACp_OI_object, (SCL, (HID, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_TermFile_CL_INIT__init_stdout__SACt_TermFile__TermFile, void,  SAC_ND_PARAM_inout( (SACp_OI_object, (SCL, (HID, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), SACt_TermFile__TermFile));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_C99Benchmarking__printResult__SACt_C99Benchmarking__Interval, , 1, inout, SACt_Interval__Interval, (SACl_int1, (SCL, (HID, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_C99Benchmarking__printResult__SACt_C99Benchmarking__Interval, void,  SAC_ND_PARAM_inout( (SACl_int1, (SCL, (HID, (NUQ, (INT, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), SACt_Interval__Interval));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__sel__i_X__ull_S, , 3, out, ulonglong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulonglong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__sel__i_X__ull_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__sel__i_X__ul_S, , 3, out, ulong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__sel__i_X__ul_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__sel__i_X__ui_S, , 3, out, uint, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, uint, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__sel__i_X__ui_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__sel__i_X__us_S, , 3, out, ushort, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ushort, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__sel__i_X__us_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__sel__i_X__ub_S, , 3, out, ubyte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ubyte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__sel__i_X__ub_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__sel__i_X__ll_S, , 3, out, longlong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, longlong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__sel__i_X__ll_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__sel__i_X__l_S, , 3, out, long, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, long, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__sel__i_X__l_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__sel__i_X__s_S, , 3, out, short, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, short, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__sel__i_X__s_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__sel__i_X__b_S, , 3, out, byte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, byte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__sel__i_X__b_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__sel__i_X__c_S, , 3, out, unsigned char, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, unsigned char, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__sel__i_X__c_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__sel__i_X__bl_S, , 3, out, bool, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__sel__i_X__bl_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__sel__i_X__d_S, , 3, out, double, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, double, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__sel__i_X__d_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__sel__i_X__f_S, , 3, out, float, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, float, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__sel__i_X__f_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__sel__i_X__i_S, , 3, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__sel__i_X__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_1__ull_S, , 3, out, ulonglong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulonglong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_1__ull_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_X__ull_S, , 3, out, ulonglong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulonglong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_X__ull_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_1__ul_S, , 3, out, ulong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_1__ul_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_X__ul_S, , 3, out, ulong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_X__ul_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_1__ui_S, , 3, out, uint, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, uint, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_1__ui_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_X__ui_S, , 3, out, uint, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, uint, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_X__ui_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_1__us_S, , 3, out, ushort, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ushort, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_1__us_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_X__us_S, , 3, out, ushort, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ushort, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_X__us_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_1__ub_S, , 3, out, ubyte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ubyte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_1__ub_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_X__ub_S, , 3, out, ubyte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ubyte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_X__ub_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_1__ll_S, , 3, out, longlong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, longlong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_1__ll_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_X__ll_S, , 3, out, longlong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, longlong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_X__ll_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_1__l_S, , 3, out, long, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, long, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_1__l_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_X__l_S, , 3, out, long, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, long, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_X__l_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_1__s_S, , 3, out, short, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, short, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_1__s_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_X__s_S, , 3, out, short, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, short, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_X__s_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_1__b_S, , 3, out, byte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, byte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_1__b_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_X__b_S, , 3, out, byte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, byte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_X__b_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_1__c_S, , 3, out, unsigned char, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, unsigned char, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_1__c_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_X__c_S, , 3, out, unsigned char, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, unsigned char, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_X__c_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_1__bl_S, , 3, out, bool, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_1__bl_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_X__bl_S, , 3, out, bool, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_X__bl_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_1__d_S, , 3, out, double, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, double, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_1__d_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_X__d_S, , 3, out, double, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, double, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_X__d_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_1__f_S, , 3, out, float, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, float, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_1__f_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_X__f_S, , 3, out, float, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, float, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_X__f_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_1__i_S, , 3, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_1__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_idx, (AKS, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selVxADistmemLocal__i_X__i_S, , 3, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selVxADistmemLocal__i_X__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_idx, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selSxADistmemLocal__i__ull_S, , 3, out, ulonglong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulonglong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selSxADistmemLocal__i__ull_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selSxADistmemLocal__i__ul_S, , 3, out, ulong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ulong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selSxADistmemLocal__i__ul_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selSxADistmemLocal__i__ui_S, , 3, out, uint, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, uint, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selSxADistmemLocal__i__ui_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selSxADistmemLocal__i__us_S, , 3, out, ushort, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ushort, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selSxADistmemLocal__i__us_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selSxADistmemLocal__i__ub_S, , 3, out, ubyte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, ubyte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selSxADistmemLocal__i__ub_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selSxADistmemLocal__i__ll_S, , 3, out, longlong, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, longlong, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selSxADistmemLocal__i__ll_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selSxADistmemLocal__i__l_S, , 3, out, long, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, long, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selSxADistmemLocal__i__l_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selSxADistmemLocal__i__s_S, , 3, out, short, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, short, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selSxADistmemLocal__i__s_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selSxADistmemLocal__i__b_S, , 3, out, byte, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, byte, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selSxADistmemLocal__i__b_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selSxADistmemLocal__i__c_S, , 3, out, unsigned char, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, unsigned char, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selSxADistmemLocal__i__c_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selSxADistmemLocal__i__bl_S, , 3, out, bool, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selSxADistmemLocal__i__bl_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selSxADistmemLocal__i__d_S, , 3, out, double, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, double, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selSxADistmemLocal__i__d_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selSxADistmemLocal__i__f_S, , 3, out, float, (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, float, (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selSxADistmemLocal__i__f_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___selSxADistmemLocal__i__i_S, , 3, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___selSxADistmemLocal__i__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_idx, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_array, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__eq__ull_S__ull_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, ulonglong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, ulonglong, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__eq__ull_S__ull_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__eq__ul_S__ul_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, ulong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, ulong, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__eq__ul_S__ul_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__eq__ui_S__ui_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, uint, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, uint, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__eq__ui_S__ui_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__eq__us_S__us_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, ushort, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, ushort, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__eq__us_S__us_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__eq__ub_S__ub_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, ubyte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, ubyte, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__eq__ub_S__ub_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__eq__ll_S__ll_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, longlong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, longlong, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__eq__ll_S__ll_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__eq__l_S__l_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, long, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, long, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__eq__l_S__l_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__eq__s_S__s_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, short, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, short, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__eq__s_S__s_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__eq__b_S__b_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, byte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, byte, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__eq__b_S__b_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__eq__c_S__c_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, unsigned char, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, unsigned char, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__eq__c_S__c_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__eq__bl_S__bl_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__eq__bl_S__bl_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__eq__d_S__d_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, double, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, double, (SACl_B, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__eq__d_S__d_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__eq__f_S__f_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, float, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_B, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__eq__f_S__f_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__eq__i_S__i_S, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__eq__i_S__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_B, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__adjustLacFunParams__bl_S__i_S__i_X, , 4, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_p, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_i, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_iv, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__adjustLacFunParams__bl_S__i_S__i_X, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_p, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_i, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_iv, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__adjustLacFunParamsReshape__bl_S__i_S__i_X__i_X, , 5, out, int, (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, bool, (SACl_p, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_i, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_iv, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_shp, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__adjustLacFunParamsReshape__bl_S__i_S__i_X__i_X, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_p, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_i, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_iv, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_shp, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__isPartitionIntersectNull__i_X__i_X__i_X__i_X, , 5, out, bool, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idxmin, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idxmax, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__isPartitionIntersectNull__i_X__i_X__i_X__i_X, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_idxmin, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_idxmax, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_bound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_bound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__isPartitionIntersectNull__i__i__i_X__i_X, , 5, out, bool, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idxmin, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idxmax, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__isPartitionIntersectNull__i__i__i_X__i_X, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_idxmin, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_idxmax, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_bound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_bound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__partitionMin__i__i, , 3, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_x, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_y, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__partitionMin__i__i, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_x, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_y, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__partitionMax__i__i, , 3, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_x, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_y, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__partitionMax__i__i, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_x, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_y, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__isPartitionIntersect1Part__i_X__i_X__i_X__i_X, , 5, out, bool, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idxmin, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idxmax, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__isPartitionIntersect1Part__i_X__i_X__i_X__i_X, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_idxmin, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_idxmax, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_bound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_bound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__isPartitionIntersect1Part__i__i__i_X__i_X, , 5, out, bool, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, int, (SACl_idxmin, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_idxmax, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_bound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__isPartitionIntersect1Part__i__i__i_X__i_X, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_idxmin, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_idxmax, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_bound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_bound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__partitionIntersectMax__i_X__i_X, , 3, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_PWLbound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ivmin, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__partitionIntersectMax__i_X__i_X, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_PWLbound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_ivmin, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__partitionIntersectMax__i_X__i, , 3, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_PWLbound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ivmin, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__partitionIntersectMax__i_X__i, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_PWLbound1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_ivmin, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__partitionIntersectMin__i_X__i_X, , 3, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_PWLbound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ivmax, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__partitionIntersectMin__i_X__i_X, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_PWLbound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_ivmax, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__partitionIntersectMin__i_X__i, , 3, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_PWLbound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ivmax, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__partitionIntersectMin__i_X__i, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_PWLbound2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_ivmax, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__prod__i_X, , 2, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_v, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__prod__i_X, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_v, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__partitionSlicer__i_X__i_X__i__i_X__i_X, , 9, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_3, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_4, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_min, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_max, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_axis, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_lb, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ub, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__partitionSlicer__i_X__i_X__i__i_X__i_X, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out( (SAC_arg_2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out( (SAC_arg_3, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out( (SAC_arg_4, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_min, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_max, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_axis, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_lb, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_ub, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__gridFiller__i_X__i_X__i_X__i__i_X, , 8, out, int, (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), out, int, (SAC_arg_3, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_lb, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_ub, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_wdth, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_dim, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_maxwidth, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__gridFiller__i_X__i_X__i_X__i__i_X, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out( (SAC_arg_2, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_out( (SAC_arg_3, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_lb, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_ub, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_wdth, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_dim, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_maxwidth, (AKD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___PL_PL__i, , 2, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_a, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___PL_PL__i, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_a, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf__MAIN_CLsacprelude_p__zero__f_X_X_X, , 2, out, float, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_A, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf__MAIN_CLsacprelude_p__zero__f_X_X_X, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_A, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_ScalarArith___PL__f__f, , 3, out, float, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_A, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_B, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_ScalarArith___PL__f__f, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_A, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_B, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p___ST__i__i, , 3, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_a, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p___ST__i__i, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_a, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));

SAC_C_EXTERN 
/*
 * ND_FUN_DECL( SACf_sacprelude_p__and__bl__bl, , 3, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_a, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DECL_FUN2( SACf_sacprelude_p__and__bl__bl, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_a, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool));

SAC_C_EXTERN 
/*
 * CUDA_GLOBALFUN_DECL( SACf__MAIN___cuknl_6985_CUDA__i__i__i__i__i__i__fd_500_26_72__f, 8, in, float, (SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0, inout, float, (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 3, in, int, (SACp_ub_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0)
 */
__global__ void SACf__MAIN___cuknl_6985_CUDA__i__i__i__i__i__i__fd_500_26_72__f(SAC_CUDA_PARAM_in( (SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_CUDA_PARAM_inout( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), float), int SAC_ND_A_MIRROR_SHAPE((SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 0), int SAC_ND_A_MIRROR_SHAPE((SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 1), int SAC_ND_A_MIRROR_SHAPE((SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 2), int SAC_ND_A_MIRROR_SIZE((SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, ))))))))))), int SAC_ND_A_MIRROR_DIM((SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, ))))))))))), SAC_CUDA_PARAM_in( (SACp_ub_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_ub_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_ub_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_lb_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_lb_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_lb_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));
;

SAC_C_EXTERN 
/*
 * CUDA_GLOBALFUN_DECL( SACf__MAIN___cuknl_6984_CUDA__i__i__i__i__i__i__fd_500_26_72__f, 8, in, float, (SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0, inout, float, (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 3, in, int, (SACp_ub_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0)
 */
__global__ void SACf__MAIN___cuknl_6984_CUDA__i__i__i__i__i__i__fd_500_26_72__f(SAC_CUDA_PARAM_in( (SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_CUDA_PARAM_inout( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), float), int SAC_ND_A_MIRROR_SHAPE((SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 0), int SAC_ND_A_MIRROR_SHAPE((SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 1), int SAC_ND_A_MIRROR_SHAPE((SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 2), int SAC_ND_A_MIRROR_SIZE((SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, ))))))))))), int SAC_ND_A_MIRROR_DIM((SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, ))))))))))), SAC_CUDA_PARAM_in( (SACp_ub_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_ub_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_ub_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_lb_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_lb_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_lb_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int));
;



/*
 *  global objects
 */

/*
 * ND_OBJDEF_EXTERN( (RandomGen, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_Random__Random, 0)
 */
SAC_ND_DECL__DATA( (RandomGen, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_Random__Random, extern)
SAC_ND_DECL__DESC( (RandomGen, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), extern)
SAC_NOTHING()

/*
 * ND_OBJDEF_EXTERN( (SACo_World__TheWorld, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_World__World, 0)
 */
SAC_ND_DECL__DATA( (SACo_World__TheWorld, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_World__World, extern)
SAC_ND_DECL__DESC( (SACo_World__TheWorld, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), extern)
SAC_NOTHING()

/*
 * ND_OBJDEF_EXTERN( (SACo_C99Benchmarking__TheBenchmarkObject, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_C99Benchmarking__C99Benchmarking, 0)
 */
SAC_ND_DECL__DATA( (SACo_C99Benchmarking__TheBenchmarkObject, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_C99Benchmarking__C99Benchmarking, extern)
SAC_ND_DECL__DESC( (SACo_C99Benchmarking__TheBenchmarkObject, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), extern)
SAC_NOTHING()

/*
 * ND_OBJDEF_EXTERN( (SACo_MTClock__TheMTClock, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_MTClock__MTClock, 0)
 */
SAC_ND_DECL__DATA( (SACo_MTClock__TheMTClock, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_MTClock__MTClock, extern)
SAC_ND_DECL__DESC( (SACo_MTClock__TheMTClock, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), extern)
SAC_NOTHING()

/*
 * ND_OBJDEF_EXTERN( (SACo_Terminal__TheTerminal, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_Terminal__Terminal, 0)
 */
SAC_ND_DECL__DATA( (SACo_Terminal__TheTerminal, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_Terminal__Terminal, extern)
SAC_ND_DECL__DESC( (SACo_Terminal__TheTerminal, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), extern)
SAC_NOTHING()

/*
 * ND_OBJDEF_EXTERN( (SACo_TermFile__stdout, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_TermFile__TermFile, 0)
 */
SAC_ND_DECL__DATA( (SACo_TermFile__stdout, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_TermFile__TermFile, extern)
SAC_ND_DECL__DESC( (SACo_TermFile__stdout, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), extern)
SAC_NOTHING()



/*
 *  function definitions (FUNDEFS)
 */



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN::SACwf__MAIN__main(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACwf__MAIN__main, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_cwc_277, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_cwc_277, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_cwc_277, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()


    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf__MAIN__main, , 1, out, int, SAC_SET_NT_USG( FAG, (SACp_cwc_277, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf__MAIN__main,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACp_cwc_277, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int))

    /*
     * ND_REFRESH__MIRROR( (SACp_cwc_277, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_NOOP()

    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_cwc_277, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_cwc_277, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACwf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN:sacprelude_p::SACwf__MAIN_CLsacprelude_p__zero__i_S(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN_CLsacprelude_p__zero__i_S, , 2, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACwf__MAIN_CLsacprelude_p__zero__i_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_cwc_356, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_cwc_356, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_cwc_356, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()


    /*
     * ND_DECL__MIRROR_PARAM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), -2)
     */
    int SAC_ND_A_MIRROR_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))));
    int SAC_ND_A_MIRROR_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, ))))))))))) = SAC_ND_A_DESC_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))));

    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf_sacprelude_p__zero__i_S, , 2, out, int, SAC_SET_NT_USG( FAG, (SACp_cwc_356, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_sacprelude_p__zero__i_S,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACp_cwc_356, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int), SAC_ND_ARG_in( SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, ))))))))))), int))

    /*
     * ND_REFRESH__MIRROR( (SACp_cwc_356, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_NOOP()

    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_cwc_356, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_cwc_356, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACwf__MAIN_CLsacprelude_p__zero__i_S, , 2, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), in, int, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN:sacprelude_p::SACwf__MAIN_CLsacprelude_p__zero__d_S(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN_CLsacprelude_p__zero__d_S, , 2, out, double, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, double, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACwf__MAIN_CLsacprelude_p__zero__d_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), double))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_cwc_358, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (DOU, )))))))))), double, 0)
     */
    SAC_ND_DECL__DATA( (SACp_cwc_358, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (DOU, )))))))))), double, )
    SAC_ND_DECL__DESC( (SACp_cwc_358, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (DOU, )))))))))), )
    SAC_NOTHING()


    /*
     * ND_DECL__MIRROR_PARAM( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), -2)
     */
    int SAC_ND_A_MIRROR_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))));
    int SAC_ND_A_MIRROR_DIM( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, ))))))))))) = SAC_ND_A_DESC_DIM( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))));

    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf_sacprelude_p__zero__d_S, , 2, out, double, SAC_SET_NT_USG( FAG, (SACp_cwc_358, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (DOU, ))))))))))), in, double, SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_sacprelude_p__zero__d_S,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACp_cwc_358, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (DOU, ))))))))))), double), SAC_ND_ARG_in( SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, ))))))))))), double))

    /*
     * ND_REFRESH__MIRROR( (SACp_cwc_358, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (DOU, )))))))))), 0)
     */
    SAC_NOOP()

    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), (SACp_cwc_358, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (DOU, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), (SACp_cwc_358, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (DOU, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACwf__MAIN_CLsacprelude_p__zero__d_S, , 2, out, double, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))), in, double, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (DOU, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN:sacprelude_p::SACwf__MAIN_CLsacprelude_p__zero__f_S(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN_CLsacprelude_p__zero__f_S, , 2, out, float, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACwf__MAIN_CLsacprelude_p__zero__f_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    SAC_ND_DECL_CONST__DATA((SACp_pinl_6975__emal_5991__cwc_360, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 0.0f)
    SAC_ND_DECL_CONST__DATA((SACp_pinl_6974__emal_5990__cwc_362, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 0.0f)
    /*
     * ND_DECL( (SACp_emal_5989__cwc_361, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_5989__cwc_361, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_5989__cwc_361, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_emal_5988__cwc_364, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BOO, )))))))))), bool, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_5988__cwc_364, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BOO, )))))))))), bool, )
    SAC_ND_DECL__DESC( (SACp_emal_5988__cwc_364, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BOO, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_cwc_359, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 0)
     */
    SAC_ND_DECL__DATA( (SACp_cwc_359, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_cwc_359, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    SAC_NOTHING()


    /*
     * ND_DECL__MIRROR_PARAM( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), -2)
     */
    int SAC_ND_A_MIRROR_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))));
    int SAC_ND_A_MIRROR_DIM( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, ))))))))))) = SAC_ND_A_DESC_DIM( (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))));

    SAC_INIT_LOCAL_MEM()
    SAC_ND_PRF_DIM_A__DATA((SACp_emal_5989__cwc_361, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), -2)
    SAC_ND_DEC_RC_FREE((SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 1, )
    SAC_ND_PRF_SxS__DATA((SACp_emal_5988__cwc_364, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BOO, )))))))))), SAC_ND_PRF_EQ, SAC_ND_READ((SACp_emal_5989__cwc_361, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 3)
    SAC_ND_FREE((SACp_emal_5989__cwc_361, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    if (SAC_ND_GETVAR((SACp_emal_5988__cwc_364, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BOO, )))))))))), SACp_emal_5988__cwc_364)) 
    { 
      SAC_ND_FREE((SACp_emal_5988__cwc_364, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BOO, )))))))))), )
      /*
       * ND_ASSIGN( (SACp_cwc_359, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_pinl_6974__emal_5990__cwc_362, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, )
       */
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_cwc_359, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 0), 106, "Assignment with incompatible types found!");
      SAC_NOOP()
      SAC_NOOP()
      SAC_NOOP()
      SAC_ND_ASSIGN__DATA( (SACp_cwc_359, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), (SACp_pinl_6974__emal_5990__cwc_362, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )

    }
    else
    { 
      SAC_ND_FREE((SACp_emal_5988__cwc_364, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BOO, )))))))))), )
      /*
       * ND_ASSIGN( (SACp_cwc_359, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_pinl_6975__emal_5991__cwc_360, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, )
       */
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_cwc_359, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 0), 106, "Assignment with incompatible types found!");
      SAC_NOOP()
      SAC_NOOP()
      SAC_NOOP()
      SAC_ND_ASSIGN__DATA( (SACp_cwc_359, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), (SACp_pinl_6975__emal_5991__cwc_360, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )

    }
    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), (SACp_cwc_359, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), (SACp_cwc_359, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACwf__MAIN_CLsacprelude_p__zero__f_S, , 2, out, float, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_A, (AUD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN:sacprelude_p::SACwf__MAIN_CLsacprelude_p__zero__c_S(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN_CLsacprelude_p__zero__c_S, , 2, out, unsigned char, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, unsigned char, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACwf__MAIN_CLsacprelude_p__zero__c_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), unsigned char))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_cwc_366, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), unsigned char, 0)
     */
    SAC_ND_DECL__DATA( (SACp_cwc_366, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), unsigned char, )
    SAC_ND_DECL__DESC( (SACp_cwc_366, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), )
    SAC_NOTHING()


    /*
     * ND_DECL__MIRROR_PARAM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), -2)
     */
    int SAC_ND_A_MIRROR_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))));
    int SAC_ND_A_MIRROR_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, ))))))))))) = SAC_ND_A_DESC_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))));

    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf_sacprelude_p__zero__c_S, , 2, out, unsigned char, SAC_SET_NT_USG( FAG, (SACp_cwc_366, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, ))))))))))), in, unsigned char, SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_sacprelude_p__zero__c_S,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACp_cwc_366, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, ))))))))))), unsigned char), SAC_ND_ARG_in( SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, ))))))))))), unsigned char))

    /*
     * ND_REFRESH__MIRROR( (SACp_cwc_366, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), 0)
     */
    SAC_NOOP()

    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), (SACp_cwc_366, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), (SACp_cwc_366, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACwf__MAIN_CLsacprelude_p__zero__c_S, , 2, out, unsigned char, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))), in, unsigned char, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UCH, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN:sacprelude_p::SACwf__MAIN_CLsacprelude_p__zero__bl_S(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN_CLsacprelude_p__zero__bl_S, , 2, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACwf__MAIN_CLsacprelude_p__zero__bl_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), bool))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_cwc_368, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BOO, )))))))))), bool, 0)
     */
    SAC_ND_DECL__DATA( (SACp_cwc_368, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BOO, )))))))))), bool, )
    SAC_ND_DECL__DESC( (SACp_cwc_368, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BOO, )))))))))), )
    SAC_NOTHING()


    /*
     * ND_DECL__MIRROR_PARAM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), -2)
     */
    int SAC_ND_A_MIRROR_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))));
    int SAC_ND_A_MIRROR_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, ))))))))))) = SAC_ND_A_DESC_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))));

    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf_sacprelude_p__zero__bl_S, , 2, out, bool, SAC_SET_NT_USG( FAG, (SACp_cwc_368, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BOO, ))))))))))), in, bool, SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_sacprelude_p__zero__bl_S,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACp_cwc_368, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BOO, ))))))))))), bool), SAC_ND_ARG_in( SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, ))))))))))), bool))

    /*
     * ND_REFRESH__MIRROR( (SACp_cwc_368, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BOO, )))))))))), 0)
     */
    SAC_NOOP()

    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), (SACp_cwc_368, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BOO, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), (SACp_cwc_368, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BOO, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACwf__MAIN_CLsacprelude_p__zero__bl_S, , 2, out, bool, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))), in, bool, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BOO, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN:sacprelude_p::SACwf__MAIN_CLsacprelude_p__zero__b_S(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN_CLsacprelude_p__zero__b_S, , 2, out, byte, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, byte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACwf__MAIN_CLsacprelude_p__zero__b_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), byte))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_cwc_370, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BYT, )))))))))), byte, 0)
     */
    SAC_ND_DECL__DATA( (SACp_cwc_370, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BYT, )))))))))), byte, )
    SAC_ND_DECL__DESC( (SACp_cwc_370, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BYT, )))))))))), )
    SAC_NOTHING()


    /*
     * ND_DECL__MIRROR_PARAM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), -2)
     */
    int SAC_ND_A_MIRROR_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))));
    int SAC_ND_A_MIRROR_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, ))))))))))) = SAC_ND_A_DESC_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))));

    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf_sacprelude_p__zero__b_S, , 2, out, byte, SAC_SET_NT_USG( FAG, (SACp_cwc_370, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BYT, ))))))))))), in, byte, SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_sacprelude_p__zero__b_S,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACp_cwc_370, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BYT, ))))))))))), byte), SAC_ND_ARG_in( SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, ))))))))))), byte))

    /*
     * ND_REFRESH__MIRROR( (SACp_cwc_370, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BYT, )))))))))), 0)
     */
    SAC_NOOP()

    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), (SACp_cwc_370, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BYT, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), (SACp_cwc_370, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (BYT, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACwf__MAIN_CLsacprelude_p__zero__b_S, , 2, out, byte, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))), in, byte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (BYT, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN:sacprelude_p::SACwf__MAIN_CLsacprelude_p__zero__s_S(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN_CLsacprelude_p__zero__s_S, , 2, out, short, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, short, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACwf__MAIN_CLsacprelude_p__zero__s_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), short))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_cwc_372, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (SHO, )))))))))), short, 0)
     */
    SAC_ND_DECL__DATA( (SACp_cwc_372, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (SHO, )))))))))), short, )
    SAC_ND_DECL__DESC( (SACp_cwc_372, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (SHO, )))))))))), )
    SAC_NOTHING()


    /*
     * ND_DECL__MIRROR_PARAM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), -2)
     */
    int SAC_ND_A_MIRROR_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))));
    int SAC_ND_A_MIRROR_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, ))))))))))) = SAC_ND_A_DESC_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))));

    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf_sacprelude_p__zero__s_S, , 2, out, short, SAC_SET_NT_USG( FAG, (SACp_cwc_372, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (SHO, ))))))))))), in, short, SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_sacprelude_p__zero__s_S,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACp_cwc_372, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (SHO, ))))))))))), short), SAC_ND_ARG_in( SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, ))))))))))), short))

    /*
     * ND_REFRESH__MIRROR( (SACp_cwc_372, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (SHO, )))))))))), 0)
     */
    SAC_NOOP()

    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), (SACp_cwc_372, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (SHO, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), (SACp_cwc_372, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (SHO, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACwf__MAIN_CLsacprelude_p__zero__s_S, , 2, out, short, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))), in, short, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (SHO, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN:sacprelude_p::SACwf__MAIN_CLsacprelude_p__zero__l_S(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN_CLsacprelude_p__zero__l_S, , 2, out, long, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, long, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACwf__MAIN_CLsacprelude_p__zero__l_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), long))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_cwc_374, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (LON, )))))))))), long, 0)
     */
    SAC_ND_DECL__DATA( (SACp_cwc_374, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (LON, )))))))))), long, )
    SAC_ND_DECL__DESC( (SACp_cwc_374, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (LON, )))))))))), )
    SAC_NOTHING()


    /*
     * ND_DECL__MIRROR_PARAM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), -2)
     */
    int SAC_ND_A_MIRROR_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))));
    int SAC_ND_A_MIRROR_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, ))))))))))) = SAC_ND_A_DESC_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))));

    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf_sacprelude_p__zero__l_S, , 2, out, long, SAC_SET_NT_USG( FAG, (SACp_cwc_374, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (LON, ))))))))))), in, long, SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_sacprelude_p__zero__l_S,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACp_cwc_374, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (LON, ))))))))))), long), SAC_ND_ARG_in( SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, ))))))))))), long))

    /*
     * ND_REFRESH__MIRROR( (SACp_cwc_374, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (LON, )))))))))), 0)
     */
    SAC_NOOP()

    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), (SACp_cwc_374, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (LON, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), (SACp_cwc_374, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (LON, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACwf__MAIN_CLsacprelude_p__zero__l_S, , 2, out, long, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))), in, long, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LON, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN:sacprelude_p::SACwf__MAIN_CLsacprelude_p__zero__ll_S(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN_CLsacprelude_p__zero__ll_S, , 2, out, longlong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, longlong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACwf__MAIN_CLsacprelude_p__zero__ll_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), longlong))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_cwc_376, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (LLO, )))))))))), longlong, 0)
     */
    SAC_ND_DECL__DATA( (SACp_cwc_376, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (LLO, )))))))))), longlong, )
    SAC_ND_DECL__DESC( (SACp_cwc_376, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (LLO, )))))))))), )
    SAC_NOTHING()


    /*
     * ND_DECL__MIRROR_PARAM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), -2)
     */
    int SAC_ND_A_MIRROR_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))));
    int SAC_ND_A_MIRROR_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, ))))))))))) = SAC_ND_A_DESC_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))));

    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf_sacprelude_p__zero__ll_S, , 2, out, longlong, SAC_SET_NT_USG( FAG, (SACp_cwc_376, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (LLO, ))))))))))), in, longlong, SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_sacprelude_p__zero__ll_S,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACp_cwc_376, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (LLO, ))))))))))), longlong), SAC_ND_ARG_in( SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, ))))))))))), longlong))

    /*
     * ND_REFRESH__MIRROR( (SACp_cwc_376, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (LLO, )))))))))), 0)
     */
    SAC_NOOP()

    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), (SACp_cwc_376, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (LLO, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), (SACp_cwc_376, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (LLO, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACwf__MAIN_CLsacprelude_p__zero__ll_S, , 2, out, longlong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))), in, longlong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (LLO, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN:sacprelude_p::SACwf__MAIN_CLsacprelude_p__zero__ub_S(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN_CLsacprelude_p__zero__ub_S, , 2, out, ubyte, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, ubyte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACwf__MAIN_CLsacprelude_p__zero__ub_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), ubyte))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_cwc_378, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UBY, )))))))))), ubyte, 0)
     */
    SAC_ND_DECL__DATA( (SACp_cwc_378, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UBY, )))))))))), ubyte, )
    SAC_ND_DECL__DESC( (SACp_cwc_378, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UBY, )))))))))), )
    SAC_NOTHING()


    /*
     * ND_DECL__MIRROR_PARAM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), -2)
     */
    int SAC_ND_A_MIRROR_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))));
    int SAC_ND_A_MIRROR_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, ))))))))))) = SAC_ND_A_DESC_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))));

    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf_sacprelude_p__zero__ub_S, , 2, out, ubyte, SAC_SET_NT_USG( FAG, (SACp_cwc_378, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UBY, ))))))))))), in, ubyte, SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_sacprelude_p__zero__ub_S,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACp_cwc_378, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UBY, ))))))))))), ubyte), SAC_ND_ARG_in( SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, ))))))))))), ubyte))

    /*
     * ND_REFRESH__MIRROR( (SACp_cwc_378, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UBY, )))))))))), 0)
     */
    SAC_NOOP()

    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), (SACp_cwc_378, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UBY, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), (SACp_cwc_378, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UBY, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACwf__MAIN_CLsacprelude_p__zero__ub_S, , 2, out, ubyte, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))), in, ubyte, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UBY, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN:sacprelude_p::SACwf__MAIN_CLsacprelude_p__zero__us_S(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN_CLsacprelude_p__zero__us_S, , 2, out, ushort, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, ushort, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACwf__MAIN_CLsacprelude_p__zero__us_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), ushort))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_cwc_380, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (USH, )))))))))), ushort, 0)
     */
    SAC_ND_DECL__DATA( (SACp_cwc_380, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (USH, )))))))))), ushort, )
    SAC_ND_DECL__DESC( (SACp_cwc_380, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (USH, )))))))))), )
    SAC_NOTHING()


    /*
     * ND_DECL__MIRROR_PARAM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), -2)
     */
    int SAC_ND_A_MIRROR_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))));
    int SAC_ND_A_MIRROR_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, ))))))))))) = SAC_ND_A_DESC_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))));

    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf_sacprelude_p__zero__us_S, , 2, out, ushort, SAC_SET_NT_USG( FAG, (SACp_cwc_380, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (USH, ))))))))))), in, ushort, SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_sacprelude_p__zero__us_S,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACp_cwc_380, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (USH, ))))))))))), ushort), SAC_ND_ARG_in( SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, ))))))))))), ushort))

    /*
     * ND_REFRESH__MIRROR( (SACp_cwc_380, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (USH, )))))))))), 0)
     */
    SAC_NOOP()

    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), (SACp_cwc_380, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (USH, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), (SACp_cwc_380, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (USH, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACwf__MAIN_CLsacprelude_p__zero__us_S, , 2, out, ushort, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))), in, ushort, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (USH, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN:sacprelude_p::SACwf__MAIN_CLsacprelude_p__zero__ui_S(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN_CLsacprelude_p__zero__ui_S, , 2, out, uint, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, uint, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACwf__MAIN_CLsacprelude_p__zero__ui_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), uint))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_cwc_382, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UIN, )))))))))), uint, 0)
     */
    SAC_ND_DECL__DATA( (SACp_cwc_382, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UIN, )))))))))), uint, )
    SAC_ND_DECL__DESC( (SACp_cwc_382, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UIN, )))))))))), )
    SAC_NOTHING()


    /*
     * ND_DECL__MIRROR_PARAM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), -2)
     */
    int SAC_ND_A_MIRROR_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))));
    int SAC_ND_A_MIRROR_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, ))))))))))) = SAC_ND_A_DESC_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))));

    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf_sacprelude_p__zero__ui_S, , 2, out, uint, SAC_SET_NT_USG( FAG, (SACp_cwc_382, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UIN, ))))))))))), in, uint, SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_sacprelude_p__zero__ui_S,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACp_cwc_382, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UIN, ))))))))))), uint), SAC_ND_ARG_in( SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, ))))))))))), uint))

    /*
     * ND_REFRESH__MIRROR( (SACp_cwc_382, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UIN, )))))))))), 0)
     */
    SAC_NOOP()

    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), (SACp_cwc_382, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UIN, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), (SACp_cwc_382, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UIN, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACwf__MAIN_CLsacprelude_p__zero__ui_S, , 2, out, uint, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))), in, uint, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (UIN, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN:sacprelude_p::SACwf__MAIN_CLsacprelude_p__zero__ul_S(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN_CLsacprelude_p__zero__ul_S, , 2, out, ulong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, ulong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACwf__MAIN_CLsacprelude_p__zero__ul_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), ulong))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_cwc_384, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (ULO, )))))))))), ulong, 0)
     */
    SAC_ND_DECL__DATA( (SACp_cwc_384, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (ULO, )))))))))), ulong, )
    SAC_ND_DECL__DESC( (SACp_cwc_384, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (ULO, )))))))))), )
    SAC_NOTHING()


    /*
     * ND_DECL__MIRROR_PARAM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), -2)
     */
    int SAC_ND_A_MIRROR_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))));
    int SAC_ND_A_MIRROR_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, ))))))))))) = SAC_ND_A_DESC_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))));

    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf_sacprelude_p__zero__ul_S, , 2, out, ulong, SAC_SET_NT_USG( FAG, (SACp_cwc_384, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (ULO, ))))))))))), in, ulong, SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_sacprelude_p__zero__ul_S,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACp_cwc_384, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (ULO, ))))))))))), ulong), SAC_ND_ARG_in( SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, ))))))))))), ulong))

    /*
     * ND_REFRESH__MIRROR( (SACp_cwc_384, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (ULO, )))))))))), 0)
     */
    SAC_NOOP()

    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), (SACp_cwc_384, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (ULO, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), (SACp_cwc_384, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (ULO, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACwf__MAIN_CLsacprelude_p__zero__ul_S, , 2, out, ulong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))), in, ulong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULO, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * Wrapper function:
 * WITH-loop Count: 0
 * _MAIN:sacprelude_p::SACwf__MAIN_CLsacprelude_p__zero__ull_S(...) [ wrapper ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACwf__MAIN_CLsacprelude_p__zero__ull_S, , 2, out, ulonglong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, ulonglong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACwf__MAIN_CLsacprelude_p__zero__ull_S, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong), SAC_ND_PARAM_in( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), ulonglong))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_cwc_386, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (ULL, )))))))))), ulonglong, 0)
     */
    SAC_ND_DECL__DATA( (SACp_cwc_386, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (ULL, )))))))))), ulonglong, )
    SAC_ND_DECL__DESC( (SACp_cwc_386, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (ULL, )))))))))), )
    SAC_NOTHING()


    /*
     * ND_DECL__MIRROR_PARAM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), -2)
     */
    int SAC_ND_A_MIRROR_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))));
    int SAC_ND_A_MIRROR_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, ))))))))))) = SAC_ND_A_DESC_DIM( (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))));

    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf_sacprelude_p__zero__ull_S, , 2, out, ulonglong, SAC_SET_NT_USG( FAG, (SACp_cwc_386, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (ULL, ))))))))))), in, ulonglong, SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_sacprelude_p__zero__ull_S,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACp_cwc_386, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (ULL, ))))))))))), ulonglong), SAC_ND_ARG_in( SAC_SET_NT_USG( FPA, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, ))))))))))), ulonglong))

    /*
     * ND_REFRESH__MIRROR( (SACp_cwc_386, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (ULL, )))))))))), 0)
     */
    SAC_NOOP()

    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), (SACp_cwc_386, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (ULL, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), (SACp_cwc_386, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (ULL, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACwf__MAIN_CLsacprelude_p__zero__ull_S, , 2, out, ulonglong, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))), in, ulonglong, (SACl_A, (AUD, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (ULL, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * WITH-loop Count: 0
 * _MAIN::SACf__MAIN__tbmm__f_X_X_X__f_X_X_X(...) [ body ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACf__MAIN__tbmm__f_X_X_X__f_X_X_X, , 3, out, float, (SAC_arg_1, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACf__MAIN__tbmm__f_X_X_X__f_X_X_X, void,  SAC_ND_PARAM_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 1, 3)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 3;
    const int SAC_ND_A_MIRROR_SIZE( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 3;
    const int SAC_ND_A_MIRROR_DIM( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;

    /*
     * ND_DECL( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 1, 3)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 3;
    const int SAC_ND_A_MIRROR_SIZE( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 3;
    const int SAC_ND_A_MIRROR_DIM( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;

    SAC_ND_DECL_CONST__DATA((SACp_emal_6026__flat_128, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 1)
    /*
     * ND_DECL( (SACp_emal_6025__flat_126, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6025__flat_126, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_6025__flat_126, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_emal_6024__flat_122, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6024__flat_122, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_6024__flat_122, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    SAC_ND_DECL_CONST__DATA((SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
    /*
     * ND_DECL( (SACp_emal_6022__flat_118, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6022__flat_118, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_6022__flat_118, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    SAC_ND_DECL_CONST__DATA((SACp_emal_6019__flat_86, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 2)
    /*
     * ND_DECL( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    SAC_ND_DECL_CONST__DATA((SACp_emal_6017__flat_70, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 0.0f)
    /*
     * ND_DECL( (SACp_emal_6016__flat_52, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6016__flat_52, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_6016__flat_52, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    SAC_ND_DECL_CONST__DATA((SACp_emal_6015__pinl_555__flat_1458, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 1.0f)
    /*
     * ND_DECL( (SACp_emal_6013__ivesli_5754, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6013__ivesli_5754, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_6013__ivesli_5754, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_emal_6011__ivesli_5756, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6011__ivesli_5756, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_6011__ivesli_5756, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_emal_6009__pinl_595__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6009__pinl_595__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_emal_6009__pinl_595__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, -3)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);
    int SAC_ND_A_MIRROR_SIZE( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))));
    const int SAC_ND_A_MIRROR_DIM( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = 1;

    /*
     * ND_DECL( (SACp_emal_6007__ivesli_5749, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6007__ivesli_5749, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_6007__ivesli_5749, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_emal_6005__ivesli_5751, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6005__ivesli_5751, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_6005__ivesli_5751, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_emal_6003__pinl_608__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6003__pinl_608__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_emal_6003__pinl_608__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, -3)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);
    int SAC_ND_A_MIRROR_SIZE( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))));
    const int SAC_ND_A_MIRROR_DIM( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = 1;

    /*
     * ND_DECL( (SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_emal_5998__pinl_619__flat_1462, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_5998__pinl_619__flat_1462, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_emal_5998__pinl_619__flat_1462, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_emal_5995__ivesli_5745, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_5995__ivesli_5745, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_5995__ivesli_5745, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_emal_5994__pinl_634__flat_3951, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_5994__pinl_634__flat_3951, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_emal_5994__pinl_634__flat_3951, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, -5)
     */
    SAC_ND_DECL__DATA( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);
    int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1);
    int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2);
    int SAC_ND_A_MIRROR_SIZE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))));
    const int SAC_ND_A_MIRROR_DIM( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = 3;

    /*
     * ND_DECL( (SACp_wlidx_5686__pinl_621_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_wlidx_5686__pinl_621_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_wlidx_5686__pinl_621_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_wlidx_5685__flat_133, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_wlidx_5685__flat_133, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_wlidx_5685__flat_133, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_wlidx_5684__flat_169, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_wlidx_5684__flat_169, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_wlidx_5684__flat_169, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_wlidx_5683_Z, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_wlidx_5683_Z, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_wlidx_5683_Z, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_pinl_622__eat_269, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_pinl_622__eat_269, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_pinl_622__eat_269, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_pinl_617_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 1, 1)
     */
    SAC_ND_DECL__DATA( (SACp_pinl_617_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_pinl_617_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_pinl_617_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 1;
    const int SAC_ND_A_MIRROR_SIZE( (SACp_pinl_617_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;
    const int SAC_ND_A_MIRROR_DIM( (SACp_pinl_617_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;

    /*
     * ND_DECL( (SACp_pinl_636__eat_268, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_pinl_636__eat_268, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_pinl_636__eat_268, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 0)
     */
    SAC_ND_DECL__DATA( (SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_pinl_633_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 1, 1)
     */
    SAC_ND_DECL__DATA( (SACp_pinl_633_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_pinl_633_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_pinl_633_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 1;
    const int SAC_ND_A_MIRROR_SIZE( (SACp_pinl_633_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;
    const int SAC_ND_A_MIRROR_DIM( (SACp_pinl_633_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;

    /*
     * ND_DECL( (SACp_eat_264, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_eat_264, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_eat_264, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_eat_263, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_eat_263, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_eat_263, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_hd_2_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 1, 1)
     */
    SAC_ND_DECL__DATA( (SACp_hd_2_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_hd_2_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_hd_2_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 1;
    const int SAC_ND_A_MIRROR_SIZE( (SACp_hd_2_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;
    const int SAC_ND_A_MIRROR_DIM( (SACp_hd_2_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;

    /*
     * ND_DECL( (SACp_hd_5_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 1, 1)
     */
    SAC_ND_DECL__DATA( (SACp_hd_5_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_hd_5_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_hd_5_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 1;
    const int SAC_ND_A_MIRROR_SIZE( (SACp_hd_5_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;
    const int SAC_ND_A_MIRROR_DIM( (SACp_hd_5_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;

    /*
     * ND_DECL( (SACl_k, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACl_k, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACl_k, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACl_n, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACl_n, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACl_n, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 1, 3)
     */
    SAC_ND_DECL__DATA( (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 3;
    const int SAC_ND_A_MIRROR_SIZE( (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 3;
    const int SAC_ND_A_MIRROR_DIM( (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;


    /*
     * ND_DECL__MIRROR_PARAM( (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), -5)
     */
    int SAC_ND_A_MIRROR_SHAPE( (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0) = SAC_ND_A_DESC_SHAPE( (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0);
    int SAC_ND_A_MIRROR_SHAPE( (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 1) = SAC_ND_A_DESC_SHAPE( (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 1);
    int SAC_ND_A_MIRROR_SHAPE( (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 2) = SAC_ND_A_DESC_SHAPE( (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 2);
    int SAC_ND_A_MIRROR_SIZE( (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))));
    const int SAC_ND_A_MIRROR_DIM( (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, ))))))))))) = 3;

    /*
     * ND_DECL__MIRROR_PARAM( (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), -5)
     */
    int SAC_ND_A_MIRROR_SHAPE( (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0) = SAC_ND_A_DESC_SHAPE( (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0);
    int SAC_ND_A_MIRROR_SHAPE( (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 1) = SAC_ND_A_DESC_SHAPE( (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 1);
    int SAC_ND_A_MIRROR_SHAPE( (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 2) = SAC_ND_A_DESC_SHAPE( (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 2);
    int SAC_ND_A_MIRROR_SIZE( (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))));
    const int SAC_ND_A_MIRROR_DIM( (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, ))))))))))) = 3;

    SAC_INIT_LOCAL_MEM()
    SAC_ND_ALLOC_BEGIN((SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 1, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 3)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 1), 19, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) == 3), 19, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 1, int)
    /*
     * ND_PRF_SHAPE_A__DATA( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), -5)
     */
    SAC_TR_PRF_PRINT( ("ND_PRF_SHAPE_A__DATA( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), -5)"))
    SAC_ND_WRITE_COPY( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_A_SHAPE( (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0), );
    SAC_ND_WRITE_COPY( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, SAC_ND_A_SHAPE( (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 1), );
    SAC_ND_WRITE_COPY( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 2, SAC_ND_A_SHAPE( (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 2), );

    SAC_ND_ALLOC_BEGIN((SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 1, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 3)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 1), 19, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) == 3), 19, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 1, int)
    /*
     * ND_PRF_SHAPE_A__DATA( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), -5)
     */
    SAC_TR_PRF_PRINT( ("ND_PRF_SHAPE_A__DATA( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), -5)"))
    SAC_ND_WRITE_COPY( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_A_SHAPE( (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0), );
    SAC_ND_WRITE_COPY( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, SAC_ND_A_SHAPE( (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 1), );
    SAC_ND_WRITE_COPY( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 2, SAC_ND_A_SHAPE( (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 2), );

    /*
     * ND_PRF_IDX_SEL__DATA( (SACp_emal_6025__flat_126, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_emal_6026__flat_128, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
     */
    SAC_TR_PRF_PRINT( ("ND_PRF_IDX_SEL__DATA( (SACp_emal_6025__flat_126, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_emal_6026__flat_128, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))"))
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6026__flat_128, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 22, "1st argument of _idx_sel_ is not a scalar!");
    SAC_ND_WRITE_READ_COPY( (SACp_emal_6025__flat_126, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), SAC_ND_READ( (SACp_emal_6026__flat_128, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), )

    /*
     * ND_PRF_IDX_SEL__DATA( (SACp_emal_6024__flat_122, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_emal_6026__flat_128, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
     */
    SAC_TR_PRF_PRINT( ("ND_PRF_IDX_SEL__DATA( (SACp_emal_6024__flat_122, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_emal_6026__flat_128, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))"))
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6026__flat_128, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 22, "1st argument of _idx_sel_ is not a scalar!");
    SAC_ND_WRITE_READ_COPY( (SACp_emal_6024__flat_122, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), SAC_ND_READ( (SACp_emal_6026__flat_128, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), )

    SAC_ND_FREE((SACp_emal_6026__flat_128, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    /*
     * ND_PRF_IDX_SEL__DATA( (SACp_emal_6022__flat_118, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
     */
    SAC_TR_PRF_PRINT( ("ND_PRF_IDX_SEL__DATA( (SACp_emal_6022__flat_118, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))"))
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 22, "1st argument of _idx_sel_ is not a scalar!");
    SAC_ND_WRITE_READ_COPY( (SACp_emal_6022__flat_118, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), SAC_ND_READ( (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), )

    SAC_ND_ALLOC_BEGIN((SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 22, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * ND_PRF_IDX_SEL__DATA( (SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
     */
    SAC_TR_PRF_PRINT( ("ND_PRF_IDX_SEL__DATA( (SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))"))
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 22, "1st argument of _idx_sel_ is not a scalar!");
    SAC_ND_WRITE_READ_COPY( (SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), SAC_ND_READ( (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), )

    SAC_ND_PRF_SxS__DATA((SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), SAC_ND_PRF_MIN, SAC_ND_READ((SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ((SACp_emal_6022__flat_118, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
    SAC_ND_FREE((SACp_emal_6022__flat_118, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    /*
     * ND_PRF_IDX_SEL__DATA( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_emal_6019__flat_86, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
     */
    SAC_TR_PRF_PRINT( ("ND_PRF_IDX_SEL__DATA( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_emal_6019__flat_86, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))"))
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6019__flat_86, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 22, "1st argument of _idx_sel_ is not a scalar!");
    SAC_ND_WRITE_READ_COPY( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), SAC_ND_READ( (SACp_emal_6019__flat_86, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), )

    /*
     * ND_PRF_IDX_SEL__DATA( (SACp_emal_6016__flat_52, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_emal_6019__flat_86, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
     */
    SAC_TR_PRF_PRINT( ("ND_PRF_IDX_SEL__DATA( (SACp_emal_6016__flat_52, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_emal_6019__flat_86, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))"))
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6019__flat_86, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 22, "1st argument of _idx_sel_ is not a scalar!");
    SAC_ND_WRITE_READ_COPY( (SACp_emal_6016__flat_52, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), SAC_ND_READ( (SACp_emal_6019__flat_86, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), )

    SAC_ND_FREE((SACp_emal_6019__flat_86, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_ALLOC_BEGIN((SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 3, float)
    /*
     * ND_SET__SHAPE_arr( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 3, SAC_ND_READ( (SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ( (SACp_emal_6024__flat_122, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ( (SACp_emal_6025__flat_126, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 3), 22, "Assignment with incompatible types found!");
    {

      int SAC_size = 1;
      SAC_size *= 
      SAC_ND_A_MIRROR_SHAPE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = 
      SAC_ND_A_DESC_SHAPE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = SAC_ND_READ( (SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);
      SAC_size *= 
      SAC_ND_A_MIRROR_SHAPE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1) = 
      SAC_ND_A_DESC_SHAPE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1) = SAC_ND_READ( (SACp_emal_6024__flat_122, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);
      SAC_size *= 
      SAC_ND_A_MIRROR_SHAPE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2) = 
      SAC_ND_A_DESC_SHAPE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2) = SAC_ND_READ( (SACp_emal_6025__flat_126, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);
      SAC_ND_A_DESC_SIZE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = SAC_ND_A_MIRROR_SIZE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = SAC_size;
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_MIRROR_SIZE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) >= 0), 22, "Array with size <0 found!");
    }

    SAC_ND_ALLOC_END((SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 3, float)
    SAC_ND_ALLOC_BEGIN((SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * ND_SET__SHAPE_arr( (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 22, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    SAC_ND_ALLOC_BEGIN((SACl_n, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * ND_SET__SHAPE_arr( (SACl_n, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACl_n, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 22, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACl_n, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    SAC_ND_ALLOC_BEGIN((SACl_k, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * ND_SET__SHAPE_arr( (SACl_k, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACl_k, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 22, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACl_k, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    SAC_ND_ALLOC_BEGIN((SACp_wlidx_5683_Z, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_wlidx_5683_Z, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_wlidx_5683_Z, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 22, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_wlidx_5683_Z, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    SAC_PF_BEGIN_WITH(genarray)
    /*
     * WL_SCHEDULE__BEGIN( 3)
     */
    {
      int SAC_WL_MT_SCHEDULE_START( 0);
      int SAC_WL_MT_SCHEDULE_STOP( 0);
      int SAC_WL_MT_SCHEDULE_START( 1);
      int SAC_WL_MT_SCHEDULE_STOP( 1);
      int SAC_WL_MT_SCHEDULE_START( 2);
      int SAC_WL_MT_SCHEDULE_STOP( 2);

      /*
       * WL_DECLARE_SHAPE_FACTOR( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -5, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3)
       */
      int SAC_WL_SHAPE_FACTOR( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);
      int SAC_WL_SHAPE_FACTOR( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1);
      int SAC_WL_SHAPE_FACTOR( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2);

      /*
       * WL_DEFINE_SHAPE_FACTOR( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -5, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3)
       */
      {
        int SAC_i;
        SAC_WL_SHAPE_FACTOR( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = 1 * SAC_ND_A_SHAPE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1) * SAC_ND_A_SHAPE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2);
        SAC_WL_SHAPE_FACTOR( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1) = 1 * SAC_ND_A_SHAPE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2);
        SAC_WL_SHAPE_FACTOR( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2) = 1;
      }

      /*
       * MT_SCHEDULER_BEGIN( 0, 3, 0, 0, 0, SAC_ND_READ( (SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ( (SACp_emal_6024__flat_122, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ( (SACp_emal_6025__flat_126, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
       */
      SAC_WL_MT_SCHEDULE_START( 0) = 0;
      SAC_WL_MT_SCHEDULE_STOP( 0) = SAC_ND_READ( (SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);
      SAC_WL_MT_SCHEDULE_START( 1) = 0;
      SAC_WL_MT_SCHEDULE_STOP( 1) = SAC_ND_READ( (SACp_emal_6024__flat_122, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);
      SAC_WL_MT_SCHEDULE_START( 2) = 0;
      SAC_WL_MT_SCHEDULE_STOP( 2) = SAC_ND_READ( (SACp_emal_6025__flat_126, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

      /*
       * WL_INIT_OFFSET( (SACp_wlidx_5683_Z, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -5, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3)
       */
      SAC_ND_WRITE( (SACp_wlidx_5683_Z, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
        = SAC_WL_MT_SCHEDULE_START( 0) * SAC_WL_SHAPE_FACTOR( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0)
        + SAC_WL_MT_SCHEDULE_START( 1) * SAC_WL_SHAPE_FACTOR( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1)
        + SAC_WL_MT_SCHEDULE_START( 2) * SAC_WL_SHAPE_FACTOR( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2);

      SAC_WL_STRIDE_LOOP0_BEGIN(0, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_READ((SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 1)
      SAC_WL_GRID_UNROLL_BEGIN(0, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 1)
      SAC_WL_STRIDE_LOOP0_BEGIN(1, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACl_n, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_READ((SACp_emal_6024__flat_122, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 1)
      SAC_WL_GRID_UNROLL_BEGIN(1, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACl_n, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 1)
      /*
       * WL_SET_OFFSET( (SACp_wlidx_5683_Z, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 3, (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -5, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACl_n, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACl_k, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
       */
      SAC_ND_WRITE( (SACp_wlidx_5683_Z, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) 
        = ( SAC_ND_A_SHAPE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2) * ( SAC_ND_A_SHAPE( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1) * SAC_ND_READ( (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
        + SAC_ND_READ( (SACl_n, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) ) + SAC_WL_MT_SCHEDULE_START( 2) ) * SAC_WL_SHAPE_FACTOR( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2);

      SAC_WL_STRIDE_LOOP0_BEGIN(2, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACl_k, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_READ((SACp_emal_6025__flat_126, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 1)
      SAC_WL_GRID_UNROLL_BEGIN(2, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACl_k, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 1)
      SAC_ND_INC_RC((SACp_emal_6017__flat_70, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1)
      SAC_ND_ALLOC_BEGIN((SACp_emal_6013__ivesli_5754, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      /*
       * ND_SET__SHAPE_arr( (SACp_emal_6013__ivesli_5754, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
       */
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6013__ivesli_5754, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 152, "Assignment with incompatible types found!");
      SAC_NOOP()

      SAC_ND_ALLOC_END((SACp_emal_6013__ivesli_5754, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      /*
       * ND_IDXS2OFFSET_id( (SACp_emal_6013__ivesli_5754, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACl_k, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
       */
      SAC_ND_WRITE( (SACp_emal_6013__ivesli_5754, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = ( SAC_ND_READ( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 2) * ( SAC_ND_READ( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1) * SAC_ND_READ( (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) + SAC_ND_READ( (SACl_k, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) ) + SAC_ND_READ( (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) );

      SAC_ND_ALLOC_BEGIN((SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 1, float)
      /*
       * ND_SET__SHAPE_arr( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, SAC_ND_READ( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
       */
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 1), 22, "Assignment with incompatible types found!");
      {

        int SAC_size = 1;
        SAC_size *= 
        SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = 
        SAC_ND_A_DESC_SHAPE( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = SAC_ND_READ( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);
        SAC_ND_A_DESC_SIZE( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = SAC_ND_A_MIRROR_SIZE( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = SAC_size;
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_MIRROR_SIZE( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) >= 0), 22, "Array with size <0 found!");
      }

      SAC_ND_ALLOC_END((SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 1, float)
      SAC_ND_ALLOC_BEGIN((SACp_eat_263, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      /*
       * ND_SET__SHAPE_arr( (SACp_eat_263, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
       */
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_eat_263, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 22, "Assignment with incompatible types found!");
      SAC_NOOP()

      SAC_ND_ALLOC_END((SACp_eat_263, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      SAC_ND_ALLOC_BEGIN((SACp_wlidx_5684__flat_169, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      /*
       * ND_SET__SHAPE_arr( (SACp_wlidx_5684__flat_169, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
       */
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_wlidx_5684__flat_169, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 22, "Assignment with incompatible types found!");
      SAC_NOOP()

      SAC_ND_ALLOC_END((SACp_wlidx_5684__flat_169, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      SAC_PF_BEGIN_WITH(genarray)
      /*
       * WL_SCHEDULE__BEGIN( 1)
       */
      {
        int SAC_WL_MT_SCHEDULE_START( 0);
        int SAC_WL_MT_SCHEDULE_STOP( 0);

        /*
         * WL_DECLARE_SHAPE_FACTOR( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_hd_5_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1)
         */
        int SAC_WL_SHAPE_FACTOR( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);

        /*
         * WL_DEFINE_SHAPE_FACTOR( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_hd_5_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1)
         */
        {
          int SAC_i;
          SAC_WL_SHAPE_FACTOR( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = 1;
        }

        /*
         * MT_SCHEDULER_BEGIN( 0, 1, 0, SAC_ND_READ( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
         */
        SAC_WL_MT_SCHEDULE_START( 0) = 0;
        SAC_WL_MT_SCHEDULE_STOP( 0) = SAC_ND_READ( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

        /*
         * WL_INIT_OFFSET( (SACp_wlidx_5684__flat_169, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_hd_5_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1)
         */
        SAC_ND_WRITE( (SACp_wlidx_5684__flat_169, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
          = SAC_WL_MT_SCHEDULE_START( 0) * SAC_WL_SHAPE_FACTOR( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);

        SAC_WL_STRIDE_LOOP0_BEGIN(0, (SACp_hd_5_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_eat_263, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_READ((SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 1)
        SAC_WL_GRID_UNROLL_BEGIN(0, (SACp_hd_5_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_eat_263, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 1)
        SAC_ND_ALLOC_BEGIN((SACp_emal_6011__ivesli_5756, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
        /*
         * ND_SET__SHAPE_arr( (SACp_emal_6011__ivesli_5756, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
         */
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6011__ivesli_5756, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 152, "Assignment with incompatible types found!");
        SAC_NOOP()

        SAC_ND_ALLOC_END((SACp_emal_6011__ivesli_5756, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
        /*
         * ND_IDXS2OFFSET_id( (SACp_emal_6011__ivesli_5756, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_eat_263, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
         */
        SAC_ND_WRITE( (SACp_emal_6011__ivesli_5756, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = ( SAC_ND_READ( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 2) * ( SAC_ND_READ( (SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1) * SAC_ND_READ( (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) + SAC_ND_READ( (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) ) + SAC_ND_READ( (SACp_eat_263, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) );

        SAC_ND_PRF_SxS__DATA((SACp_emal_6011__ivesli_5756, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), SAC_ND_PRF_ADD, SAC_ND_READ((SACp_emal_6013__ivesli_5754, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ((SACp_emal_6011__ivesli_5756, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
        SAC_ND_ALLOC_BEGIN((SACp_emal_6009__pinl_595__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 0, float)
        /*
         * ND_SET__SHAPE_arr( (SACp_emal_6009__pinl_595__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0)
         */
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6009__pinl_595__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 0), 152, "Assignment with incompatible types found!");
        SAC_NOOP()

        SAC_ND_ALLOC_END((SACp_emal_6009__pinl_595__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 0, float)
        /*
         * ND_PRF_IDX_SEL__DATA( (SACp_emal_6009__pinl_595__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), -5, (SACp_emal_6011__ivesli_5756, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
         */
        SAC_TR_PRF_PRINT( ("ND_PRF_IDX_SEL__DATA( (SACp_emal_6009__pinl_595__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), -5, (SACp_emal_6011__ivesli_5756, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))"))
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6011__ivesli_5756, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 152, "1st argument of _idx_sel_ is not a scalar!");
        SAC_ND_WRITE_READ_COPY( (SACp_emal_6009__pinl_595__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), SAC_ND_READ( (SACp_emal_6011__ivesli_5756, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), )

        SAC_ND_FREE((SACp_emal_6011__ivesli_5756, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
        /*
         * WL_ASSIGN( (SACp_emal_6009__pinl_595__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_hd_5_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_wlidx_5684__flat_169, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
         */
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6009__pinl_595__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == (SAC_ND_A_DIM( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) - SAC_ND_A_SIZE( (SACp_hd_5_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))), 22, "WL expression with illegal dimension found!");
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SIZE( (SACp_emal_6009__pinl_595__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == SAC_WL_SHAPE_FACTOR( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0)), 22, "WL expression with illegal size found!");
        SAC_ND_WRITE_READ_COPY( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), SAC_ND_READ( (SACp_wlidx_5684__flat_169, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), (SACp_emal_6009__pinl_595__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, );

        SAC_ND_FREE((SACp_emal_6009__pinl_595__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
        SAC_WL_INC_OFFSET((SACp_wlidx_5684__flat_169, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_emal_6009__pinl_595__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))))
        SAC_WL_GRID_UNROLL_END(0, (SACp_hd_5_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_eat_263, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 1)
        SAC_WL_STRIDE_LOOP_END(0, (SACp_hd_5_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_eat_263, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_READ((SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 1)
        /*
         * MT_SCHEDULER_END( 0, 1, 0, SAC_ND_READ( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
         */


        /*
         * WL_SCHEDULE__END( 1)
         */
      }

      SAC_PF_END_WITH(genarray)
      SAC_ND_LABEL(_comp_6989_SAC_label)
      SAC_ND_FREE((SACp_eat_263, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
      SAC_ND_FREE((SACp_wlidx_5684__flat_169, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
      SAC_ND_FREE((SACp_emal_6013__ivesli_5754, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
      SAC_ND_ALLOC_BEGIN((SACp_emal_6007__ivesli_5749, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      /*
       * ND_SET__SHAPE_arr( (SACp_emal_6007__ivesli_5749, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
       */
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6007__ivesli_5749, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 152, "Assignment with incompatible types found!");
      SAC_NOOP()

      SAC_ND_ALLOC_END((SACp_emal_6007__ivesli_5749, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      /*
       * ND_IDXS2OFFSET_id( (SACp_emal_6007__ivesli_5749, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACl_n, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
       */
      SAC_ND_WRITE( (SACp_emal_6007__ivesli_5749, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = ( SAC_ND_READ( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 2) * ( SAC_ND_READ( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1) * SAC_ND_READ( (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) + SAC_ND_READ( (SACl_n, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) ) + SAC_ND_READ( (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) );

      SAC_ND_ALLOC_BEGIN((SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 1, float)
      /*
       * ND_SET__SHAPE_arr( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, SAC_ND_READ( (SACp_emal_6016__flat_52, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
       */
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 1), 22, "Assignment with incompatible types found!");
      {

        int SAC_size = 1;
        SAC_size *= 
        SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = 
        SAC_ND_A_DESC_SHAPE( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = SAC_ND_READ( (SACp_emal_6016__flat_52, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);
        SAC_ND_A_DESC_SIZE( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = SAC_ND_A_MIRROR_SIZE( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = SAC_size;
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_MIRROR_SIZE( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) >= 0), 22, "Array with size <0 found!");
      }

      SAC_ND_ALLOC_END((SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 1, float)
      SAC_ND_ALLOC_BEGIN((SACp_eat_264, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      /*
       * ND_SET__SHAPE_arr( (SACp_eat_264, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
       */
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_eat_264, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 22, "Assignment with incompatible types found!");
      SAC_NOOP()

      SAC_ND_ALLOC_END((SACp_eat_264, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      SAC_ND_ALLOC_BEGIN((SACp_wlidx_5685__flat_133, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      /*
       * ND_SET__SHAPE_arr( (SACp_wlidx_5685__flat_133, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
       */
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_wlidx_5685__flat_133, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 22, "Assignment with incompatible types found!");
      SAC_NOOP()

      SAC_ND_ALLOC_END((SACp_wlidx_5685__flat_133, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      SAC_PF_BEGIN_WITH(genarray)
      /*
       * WL_SCHEDULE__BEGIN( 1)
       */
      {
        int SAC_WL_MT_SCHEDULE_START( 0);
        int SAC_WL_MT_SCHEDULE_STOP( 0);

        /*
         * WL_DECLARE_SHAPE_FACTOR( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_hd_2_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1)
         */
        int SAC_WL_SHAPE_FACTOR( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);

        /*
         * WL_DEFINE_SHAPE_FACTOR( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_hd_2_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1)
         */
        {
          int SAC_i;
          SAC_WL_SHAPE_FACTOR( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = 1;
        }

        /*
         * MT_SCHEDULER_BEGIN( 0, 1, 0, SAC_ND_READ( (SACp_emal_6016__flat_52, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
         */
        SAC_WL_MT_SCHEDULE_START( 0) = 0;
        SAC_WL_MT_SCHEDULE_STOP( 0) = SAC_ND_READ( (SACp_emal_6016__flat_52, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

        /*
         * WL_INIT_OFFSET( (SACp_wlidx_5685__flat_133, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_hd_2_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1)
         */
        SAC_ND_WRITE( (SACp_wlidx_5685__flat_133, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
          = SAC_WL_MT_SCHEDULE_START( 0) * SAC_WL_SHAPE_FACTOR( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);

        SAC_WL_STRIDE_LOOP0_BEGIN(0, (SACp_hd_2_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_eat_264, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_READ((SACp_emal_6016__flat_52, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 1)
        SAC_WL_GRID_UNROLL_BEGIN(0, (SACp_hd_2_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_eat_264, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 1)
        SAC_ND_ALLOC_BEGIN((SACp_emal_6005__ivesli_5751, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
        /*
         * ND_SET__SHAPE_arr( (SACp_emal_6005__ivesli_5751, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
         */
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6005__ivesli_5751, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 152, "Assignment with incompatible types found!");
        SAC_NOOP()

        SAC_ND_ALLOC_END((SACp_emal_6005__ivesli_5751, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
        /*
         * ND_IDXS2OFFSET_id( (SACp_emal_6005__ivesli_5751, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_eat_264, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
         */
        SAC_ND_WRITE( (SACp_emal_6005__ivesli_5751, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = ( SAC_ND_READ( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 2) * ( SAC_ND_READ( (SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1) * SAC_ND_READ( (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) + SAC_ND_READ( (SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) ) + SAC_ND_READ( (SACp_eat_264, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) );

        SAC_ND_PRF_SxS__DATA((SACp_emal_6005__ivesli_5751, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), SAC_ND_PRF_ADD, SAC_ND_READ((SACp_emal_6007__ivesli_5749, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ((SACp_emal_6005__ivesli_5751, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
        SAC_ND_ALLOC_BEGIN((SACp_emal_6003__pinl_608__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 0, float)
        /*
         * ND_SET__SHAPE_arr( (SACp_emal_6003__pinl_608__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0)
         */
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6003__pinl_608__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 0), 152, "Assignment with incompatible types found!");
        SAC_NOOP()

        SAC_ND_ALLOC_END((SACp_emal_6003__pinl_608__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 0, float)
        /*
         * ND_PRF_IDX_SEL__DATA( (SACp_emal_6003__pinl_608__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), -5, (SACp_emal_6005__ivesli_5751, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
         */
        SAC_TR_PRF_PRINT( ("ND_PRF_IDX_SEL__DATA( (SACp_emal_6003__pinl_608__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), -5, (SACp_emal_6005__ivesli_5751, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))"))
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6005__ivesli_5751, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 152, "1st argument of _idx_sel_ is not a scalar!");
        SAC_ND_WRITE_READ_COPY( (SACp_emal_6003__pinl_608__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), SAC_ND_READ( (SACp_emal_6005__ivesli_5751, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), )

        SAC_ND_FREE((SACp_emal_6005__ivesli_5751, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
        /*
         * WL_ASSIGN( (SACp_emal_6003__pinl_608__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_hd_2_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_wlidx_5685__flat_133, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
         */
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6003__pinl_608__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == (SAC_ND_A_DIM( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) - SAC_ND_A_SIZE( (SACp_hd_2_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))), 22, "WL expression with illegal dimension found!");
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SIZE( (SACp_emal_6003__pinl_608__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == SAC_WL_SHAPE_FACTOR( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0)), 22, "WL expression with illegal size found!");
        SAC_ND_WRITE_READ_COPY( (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), SAC_ND_READ( (SACp_wlidx_5685__flat_133, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), (SACp_emal_6003__pinl_608__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, );

        SAC_ND_FREE((SACp_emal_6003__pinl_608__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
        SAC_WL_INC_OFFSET((SACp_wlidx_5685__flat_133, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_emal_6003__pinl_608__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))))
        SAC_WL_GRID_UNROLL_END(0, (SACp_hd_2_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_eat_264, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 1)
        SAC_WL_STRIDE_LOOP_END(0, (SACp_hd_2_index, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_eat_264, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_READ((SACp_emal_6016__flat_52, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 1)
        /*
         * MT_SCHEDULER_END( 0, 1, 0, SAC_ND_READ( (SACp_emal_6016__flat_52, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
         */


        /*
         * WL_SCHEDULE__END( 1)
         */
      }

      SAC_PF_END_WITH(genarray)
      SAC_ND_LABEL(_comp_6990_SAC_label)
      SAC_ND_FREE((SACp_eat_264, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
      SAC_ND_FREE((SACp_wlidx_5685__flat_133, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
      SAC_ND_FREE((SACp_emal_6007__ivesli_5749, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
      SAC_ND_ALLOC_BEGIN((SACp_pinl_622__eat_269, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      /*
       * ND_SET__SHAPE_arr( (SACp_pinl_622__eat_269, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
       */
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_pinl_622__eat_269, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 149, "Assignment with incompatible types found!");
      SAC_NOOP()

      SAC_ND_ALLOC_END((SACp_pinl_622__eat_269, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      SAC_ND_ALLOC_BEGIN((SACp_wlidx_5686__pinl_621_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      /*
       * ND_SET__SHAPE_arr( (SACp_wlidx_5686__pinl_621_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
       */
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_wlidx_5686__pinl_621_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 149, "Assignment with incompatible types found!");
      SAC_NOOP()

      SAC_ND_ALLOC_END((SACp_wlidx_5686__pinl_621_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      SAC_PF_BEGIN_WITH(genarray)
      /*
       * WL_SCHEDULE__BEGIN( 1)
       */
      {
        int SAC_WL_MT_SCHEDULE_START( 0);
        int SAC_WL_MT_SCHEDULE_STOP( 0);

        /*
         * WL_DECLARE_SHAPE_FACTOR( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_pinl_617_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1)
         */
        int SAC_WL_SHAPE_FACTOR( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);

        /*
         * WL_DEFINE_SHAPE_FACTOR( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_pinl_617_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1)
         */
        {
          int SAC_i;
          SAC_WL_SHAPE_FACTOR( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = 1;
        }

        /*
         * MT_SCHEDULER_BEGIN( 0, 1, 0, SAC_ND_READ( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
         */
        SAC_WL_MT_SCHEDULE_START( 0) = 0;
        SAC_WL_MT_SCHEDULE_STOP( 0) = SAC_ND_READ( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

        /*
         * WL_INIT_OFFSET( (SACp_wlidx_5686__pinl_621_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_pinl_617_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1)
         */
        SAC_ND_WRITE( (SACp_wlidx_5686__pinl_621_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
          = SAC_WL_MT_SCHEDULE_START( 0) * SAC_WL_SHAPE_FACTOR( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);

        SAC_WL_STRIDE_LOOP0_BEGIN(0, (SACp_pinl_617_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_pinl_622__eat_269, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_READ((SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 1)
        SAC_WL_GRID_UNROLL_BEGIN(0, (SACp_pinl_617_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_pinl_622__eat_269, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 1)
        SAC_ND_ALLOC_BEGIN((SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
        /*
         * ND_SET__SHAPE_arr( (SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
         */
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 149, "Assignment with incompatible types found!");
        SAC_NOOP()

        SAC_ND_ALLOC_END((SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
        /*
         * ND_IDXS2OFFSET_arr( (SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_pinl_622__eat_269, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, SAC_ND_READ( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
         */
        SAC_ND_WRITE( (SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = SAC_ND_READ( (SACp_pinl_622__eat_269, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

        SAC_ND_ALLOC_BEGIN((SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 0, float)
        /*
         * ND_SET__SHAPE_arr( (SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0)
         */
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 0), 149, "Assignment with incompatible types found!");
        SAC_NOOP()

        SAC_ND_ALLOC_END((SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 0, float)
        /*
         * ND_PRF_IDX_SEL__DATA( (SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
         */
        SAC_TR_PRF_PRINT( ("ND_PRF_IDX_SEL__DATA( (SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))"))
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 149, "1st argument of _idx_sel_ is not a scalar!");
        SAC_ND_WRITE_READ_COPY( (SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), SAC_ND_READ( (SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), )

        SAC_ND_ALLOC_BEGIN((SACp_emal_5998__pinl_619__flat_1462, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 0, float)
        /*
         * ND_SET__SHAPE_arr( (SACp_emal_5998__pinl_619__flat_1462, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0)
         */
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_5998__pinl_619__flat_1462, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 0), 149, "Assignment with incompatible types found!");
        SAC_NOOP()

        SAC_ND_ALLOC_END((SACp_emal_5998__pinl_619__flat_1462, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 0, float)
        /*
         * ND_PRF_IDX_SEL__DATA( (SACp_emal_5998__pinl_619__flat_1462, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
         */
        SAC_TR_PRF_PRINT( ("ND_PRF_IDX_SEL__DATA( (SACp_emal_5998__pinl_619__flat_1462, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))"))
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 149, "1st argument of _idx_sel_ is not a scalar!");
        SAC_ND_WRITE_READ_COPY( (SACp_emal_5998__pinl_619__flat_1462, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), SAC_ND_READ( (SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), )

        SAC_ND_FREE((SACp_emal_6000__ivesli_5747, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
        SAC_ND_PRF_SxS__DATA((SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), SAC_ND_PRF_MUL, SAC_ND_READ((SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0), SAC_ND_READ((SACp_emal_5998__pinl_619__flat_1462, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0))
        SAC_ND_FREE((SACp_emal_5998__pinl_619__flat_1462, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
        /*
         * WL_ASSIGN( (SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_pinl_617_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_wlidx_5686__pinl_621_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
         */
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == (SAC_ND_A_DIM( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) - SAC_ND_A_SIZE( (SACp_pinl_617_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))), 1, "WL expression with illegal dimension found!");
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SIZE( (SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == SAC_WL_SHAPE_FACTOR( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0)), 1, "WL expression with illegal size found!");
        SAC_ND_WRITE_READ_COPY( (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), SAC_ND_READ( (SACp_wlidx_5686__pinl_621_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), (SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, );

        SAC_ND_FREE((SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
        SAC_WL_INC_OFFSET((SACp_wlidx_5686__pinl_621_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_emal_5999__pinl_618__flat_1463, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))))
        SAC_WL_GRID_UNROLL_END(0, (SACp_pinl_617_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_pinl_622__eat_269, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 1)
        SAC_WL_STRIDE_LOOP_END(0, (SACp_pinl_617_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_pinl_622__eat_269, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_READ((SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 1)
        /*
         * MT_SCHEDULER_END( 0, 1, 0, SAC_ND_READ( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
         */


        /*
         * WL_SCHEDULE__END( 1)
         */
      }

      SAC_PF_END_WITH(genarray)
      SAC_ND_LABEL(_comp_6991_SAC_label)
      SAC_ND_FREE((SACp_emal_6002__flat_133, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
      SAC_ND_FREE((SACp_pinl_622__eat_269, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
      SAC_ND_FREE((SACp_wlidx_5686__pinl_621_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
      SAC_ND_ALLOC_BEGIN((SACp_pinl_636__eat_268, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      /*
       * ND_SET__SHAPE_arr( (SACp_pinl_636__eat_268, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
       */
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_pinl_636__eat_268, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 629, "Assignment with incompatible types found!");
      SAC_NOOP()

      SAC_ND_ALLOC_END((SACp_pinl_636__eat_268, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
      /*
       * ND_ASSIGN( (SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_6017__flat_70, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, )
       */
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 0), 629, "Assignment with incompatible types found!");
      SAC_NOOP()
      SAC_NOOP()
      SAC_NOOP()
      SAC_ND_ASSIGN__DATA( (SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), (SACp_emal_6017__flat_70, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )

      SAC_PF_BEGIN_WITH(fold)
      /*
       * WL_SCHEDULE__BEGIN( 1)
       */
      {
        int SAC_WL_MT_SCHEDULE_START( 0);
        int SAC_WL_MT_SCHEDULE_STOP( 0);

        /*
         * MT_SCHEDULER_BEGIN( 0, 1, 0, SAC_ND_READ( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
         */
        SAC_WL_MT_SCHEDULE_START( 0) = 0;
        SAC_WL_MT_SCHEDULE_STOP( 0) = SAC_ND_READ( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

        SAC_WL_STRIDE_LOOP0_BEGIN(0, (SACp_pinl_633_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_pinl_636__eat_268, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_READ((SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 1)
        SAC_WL_GRID_UNROLL_BEGIN(0, (SACp_pinl_633_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_pinl_636__eat_268, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 1)
        SAC_NOOP()
        SAC_ND_ALLOC_BEGIN((SACp_emal_5995__ivesli_5745, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
        /*
         * ND_SET__SHAPE_arr( (SACp_emal_5995__ivesli_5745, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
         */
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_5995__ivesli_5745, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 629, "Assignment with incompatible types found!");
        SAC_NOOP()

        SAC_ND_ALLOC_END((SACp_emal_5995__ivesli_5745, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
        /*
         * ND_IDXS2OFFSET_arr( (SACp_emal_5995__ivesli_5745, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_pinl_636__eat_268, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, SAC_ND_READ( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
         */
        SAC_ND_WRITE( (SACp_emal_5995__ivesli_5745, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = SAC_ND_READ( (SACp_pinl_636__eat_268, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

        SAC_ND_ALLOC_BEGIN((SACp_emal_5994__pinl_634__flat_3951, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 0, float)
        /*
         * ND_SET__SHAPE_arr( (SACp_emal_5994__pinl_634__flat_3951, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0)
         */
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_5994__pinl_634__flat_3951, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 0), 629, "Assignment with incompatible types found!");
        SAC_NOOP()

        SAC_ND_ALLOC_END((SACp_emal_5994__pinl_634__flat_3951, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 0, float)
        /*
         * ND_PRF_IDX_SEL__DATA( (SACp_emal_5994__pinl_634__flat_3951, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_emal_5995__ivesli_5745, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
         */
        SAC_TR_PRF_PRINT( ("ND_PRF_IDX_SEL__DATA( (SACp_emal_5994__pinl_634__flat_3951, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -3, (SACp_emal_5995__ivesli_5745, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))"))
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_5995__ivesli_5745, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 629, "1st argument of _idx_sel_ is not a scalar!");
        SAC_ND_WRITE_READ_COPY( (SACp_emal_5994__pinl_634__flat_3951, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), SAC_ND_READ( (SACp_emal_5995__ivesli_5745, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), )

        SAC_ND_FREE((SACp_emal_5995__ivesli_5745, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
        SAC_ND_PRF_SxS__DATA((SACp_emal_5994__pinl_634__flat_3951, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), SAC_ND_PRF_ADD, SAC_ND_READ((SACp_emal_5994__pinl_634__flat_3951, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0), SAC_ND_READ((SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0))
        SAC_ND_DEC_RC_FREE((SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, )
        /*
         * ND_ASSIGN( (SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_5994__pinl_634__flat_3951, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, )
         */
        SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 0), 117, "Assignment with incompatible types found!");
        SAC_NOOP()
        SAC_NOOP()
        SAC_NOOP()
        SAC_ND_ASSIGN__DATA( (SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), (SACp_emal_5994__pinl_634__flat_3951, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )

        /*
         * WL_FOLD( (SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_pinl_633_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACp_pinl_636__eat_268, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
         */
        SAC_TR_WL_PRINT( ("index vector [%d] -- fold", SAC_ND_READ( (SACp_pinl_636__eat_268, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)));
        /* fold operation */

        SAC_WL_GRID_UNROLL_END(0, (SACp_pinl_633_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_pinl_636__eat_268, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 1)
        SAC_WL_STRIDE_LOOP_END(0, (SACp_pinl_633_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_pinl_636__eat_268, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_READ((SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 1)
        /*
         * MT_SCHEDULER_END( 0, 1, 0, SAC_ND_READ( (SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
         */


        /*
         * WL_SCHEDULE__END( 1)
         */
      }

      SAC_PF_END_WITH(fold)
      SAC_ND_LABEL(_comp_6992_SAC_label)
      SAC_ND_FREE((SACp_pinl_636__eat_268, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
      SAC_ND_FREE((SACp_emal_6008__flat_169, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
      /*
       * WL_ASSIGN( (SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -5, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACp_wlidx_5683_Z, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
       */
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == (SAC_ND_A_DIM( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) - SAC_ND_A_SIZE( (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))), 22, "WL expression with illegal dimension found!");
      SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SIZE( (SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == SAC_WL_SHAPE_FACTOR( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2)), 22, "WL expression with illegal size found!");
      SAC_ND_WRITE_READ_COPY( (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), SAC_ND_READ( (SACp_wlidx_5683_Z, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), (SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, );

      SAC_ND_DEC_RC_FREE((SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, )
      SAC_WL_INC_OFFSET((SACp_wlidx_5683_Z, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_pinl_635_res, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))))
      SAC_WL_GRID_UNROLL_END(2, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACl_k, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 1)
      SAC_WL_STRIDE_LOOP_END(2, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACl_k, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_READ((SACp_emal_6025__flat_126, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 1)
      SAC_WL_GRID_UNROLL_END(1, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACl_n, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 1)
      SAC_WL_STRIDE_LOOP_END(1, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACl_n, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_READ((SACp_emal_6024__flat_122, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 1)
      SAC_WL_GRID_UNROLL_END(0, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 1)
      SAC_WL_STRIDE_LOOP_END(0, (SACp_flat_130, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_READ((SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 1)
      /*
       * MT_SCHEDULER_END( 0, 3, 0, 0, 0, SAC_ND_READ( (SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ( (SACp_emal_6024__flat_122, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ( (SACp_emal_6025__flat_126, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
       */


      /*
       * WL_SCHEDULE__END( 3)
       */
    }

    SAC_PF_END_WITH(genarray)
    SAC_ND_LABEL(_comp_6993_SAC_label)
    SAC_ND_FREE((SACp_emal_6025__flat_126, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6024__flat_122, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6021__flat_114, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6023__flat_120, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6018__flat_84, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6017__flat_70, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    SAC_ND_FREE((SACp_emal_6016__flat_52, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACl_b, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACl_n, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACl_k, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6015__pinl_555__flat_1458, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    SAC_ND_FREE((SACp_emal_6028__isaa_1067_Y, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6027__isaa_1070_X, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_wlidx_5683_Z, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_DEC_RC_FREE((SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 1, )
    SAC_ND_DEC_RC_FREE((SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 1, )
    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), (SACp_emal_5992_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACf__MAIN__tbmm__f_X_X_X__f_X_X_X, , 3, out, float, (SAC_arg_1, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * WITH-loop Count: 0
 * _MAIN::SACf__MAIN__main(...) [ body ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACf__MAIN__main, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    /*
     * ND_DECL( (SACp_tcp_6988__emal_6033__flat_228, (AKD, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), unsigned char, -3)
     */
    SAC_ND_DECL__DATA( (SACp_tcp_6988__emal_6033__flat_228, (AKD, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), unsigned char, )
    SAC_ND_DECL__DESC( (SACp_tcp_6988__emal_6033__flat_228, (AKD, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), )
    int SAC_ND_A_MIRROR_SHAPE( (SACp_tcp_6988__emal_6033__flat_228, (AKD, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), 0);
    int SAC_ND_A_MIRROR_SIZE( (SACp_tcp_6988__emal_6033__flat_228, (AKD, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))));
    const int SAC_ND_A_MIRROR_DIM( (SACp_tcp_6988__emal_6033__flat_228, (AKD, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, ))))))))))) = 1;

    /*
     * ND_DECL( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, -5)
     */
    SAC_ND_DECL__DATA( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    int SAC_ND_A_MIRROR_SHAPE( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);
    int SAC_ND_A_MIRROR_SHAPE( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1);
    int SAC_ND_A_MIRROR_SHAPE( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2);
    int SAC_ND_A_MIRROR_SIZE( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))));
    const int SAC_ND_A_MIRROR_DIM( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = 3;

    /*
     * ND_DECL( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, -5)
     */
    SAC_ND_DECL__DATA( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    int SAC_ND_A_MIRROR_SHAPE( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);
    int SAC_ND_A_MIRROR_SHAPE( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1);
    int SAC_ND_A_MIRROR_SHAPE( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2);
    int SAC_ND_A_MIRROR_SIZE( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))));
    const int SAC_ND_A_MIRROR_DIM( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = 3;

    SAC_ND_DECL_CONST__DATA((SACp_emal_6059__isaa_1146__rso_249_TheWorld, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
    SAC_ND_DECL_CONST__DATA((SACp_emal_6058__flat_215, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 2147483647)
    /*
     * ND_DECL( (SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    SAC_NOTHING()

    SAC_ND_DECL_CONST__DATA((SACp_emal_6055__cnstass_5980_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
    SAC_ND_DECL_CONST__DATA((SACp_emal_6054__cnstass_5979_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
    SAC_ND_DECL_CONST__DATA((SACp_emal_6053__cnstass_5978_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
    SAC_ND_DECL_CONST__DATA((SACp_emal_6052__cnstass_5977_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 72)
    SAC_ND_DECL_CONST__DATA((SACp_emal_6051__cnstass_5976_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 26)
    SAC_ND_DECL_CONST__DATA((SACp_emal_6050__cnstass_5975_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 500)
    /*
     * ND_DECL( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), float, 3, 500, 26, 72)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 0) = 500;
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 1) = 26;
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 2) = 72;
    const int SAC_ND_A_MIRROR_SIZE( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))) = 936000;
    const int SAC_ND_A_MIRROR_DIM( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))) = 3;

    /*
     * ND_DECL( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 3, 500, 26, 72)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = 500;
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1) = 26;
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2) = 72;
    const int SAC_ND_A_MIRROR_SIZE( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = 936000;
    const int SAC_ND_A_MIRROR_DIM( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = 3;

    /*
     * ND_DECL( (SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    SAC_NOTHING()

    SAC_ND_DECL_CONST__DATA((SACp_emal_6044__cnstass_5974_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
    SAC_ND_DECL_CONST__DATA((SACp_emal_6043__cnstass_5973_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
    SAC_ND_DECL_CONST__DATA((SACp_emal_6042__cnstass_5972_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
    SAC_ND_DECL_CONST__DATA((SACp_emal_6041__cnstass_5971_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 72)
    SAC_ND_DECL_CONST__DATA((SACp_emal_6040__cnstass_5970_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 26)
    SAC_ND_DECL_CONST__DATA((SACp_emal_6039__cnstass_5969_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 500)
    /*
     * ND_DECL( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), float, 3, 500, 26, 72)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 0) = 500;
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 1) = 26;
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 2) = 72;
    const int SAC_ND_A_MIRROR_SIZE( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))) = 936000;
    const int SAC_ND_A_MIRROR_DIM( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))) = 3;

    /*
     * ND_DECL( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 3, 500, 26, 72)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = 500;
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1) = 26;
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2) = 72;
    const int SAC_ND_A_MIRROR_SIZE( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = 936000;
    const int SAC_ND_A_MIRROR_DIM( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = 3;

    SAC_ND_DECL_CONST__DATA((SACp_emal_6035__flat_236, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 2)
    SAC_ND_DECL_CONST__DATA((SACp_emal_6034__flat_234, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 4)
    /*
     * ND_DECL( (SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), unsigned char, 1, 5)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), unsigned char, )
    SAC_ND_DECL__DESC( (SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), 0) = 5;
    const int SAC_ND_A_MIRROR_SIZE( (SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, ))))))))))) = 5;
    const int SAC_ND_A_MIRROR_DIM( (SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, ))))))))))) = 1;

    /*
     * ND_DECL( (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 1, 3)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 3;
    const int SAC_ND_A_MIRROR_SIZE( (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 3;
    const int SAC_ND_A_MIRROR_DIM( (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;

    /*
     * ND_DECL( (SACp_emal_6031__ivesli_5759, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6031__ivesli_5759, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_6031__ivesli_5759, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_emal_6030__pinl_698__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6030__pinl_698__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACp_emal_6030__pinl_698__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_emal_6029__pinl_701__flat_252, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_emal_6029__pinl_701__flat_252, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_emal_6029__pinl_701__flat_252, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_wlidx_5688_Y, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_wlidx_5688_Y, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_wlidx_5688_Y, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_wlidx_5687_X, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_wlidx_5687_X, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_wlidx_5687_X, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 1, 3)
     */
    SAC_ND_DECL__DATA( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 3;
    const int SAC_ND_A_MIRROR_SIZE( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 3;
    const int SAC_ND_A_MIRROR_DIM( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;

    /*
     * ND_DECL( (SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 1, 3)
     */
    SAC_ND_DECL__DATA( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    const int SAC_ND_A_MIRROR_SHAPE( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 3;
    const int SAC_ND_A_MIRROR_SIZE( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 3;
    const int SAC_ND_A_MIRROR_DIM( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;

    /*
     * ND_DECL( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, -5)
     */
    SAC_ND_DECL__DATA( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, )
    SAC_ND_DECL__DESC( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    int SAC_ND_A_MIRROR_SHAPE( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);
    int SAC_ND_A_MIRROR_SHAPE( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1);
    int SAC_ND_A_MIRROR_SHAPE( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2);
    int SAC_ND_A_MIRROR_SIZE( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))));
    const int SAC_ND_A_MIRROR_DIM( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = 3;

    /*
     * ND_DECL( (SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_Interval__Interval, 0)
     */
    SAC_ND_DECL__DATA( (SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_Interval__Interval, )
    SAC_ND_DECL__DESC( (SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_flat_227, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_String__string, 0)
     */
    SAC_ND_DECL__DATA( (SACp_flat_227, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), SACt_String__string, )
    SAC_ND_DECL__DESC( (SACp_flat_227, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_flat_222, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_flat_222, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_flat_222, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()

    /*
     * ND_DECL( (SACp_flat_211, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
     */
    SAC_ND_DECL__DATA( (SACp_flat_211, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
    SAC_ND_DECL__DESC( (SACp_flat_211, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOTHING()


    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf__MAIN_CL_INIT__init, , 0)
     */
    SAC_ND_FUNAP2( SACf__MAIN_CL_INIT__init, )

    /*
     * ND_FUN_AP( SACrandom, SACp_flat_211, 2, in_nodesc, int, SAC_SET_NT_USG( FAG, (SACp_emal_6059__isaa_1146__rso_249_TheWorld, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in_nodesc, int, SAC_SET_NT_USG( FAG, (SACp_emal_6058__flat_215, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))
     */
    SACp_flat_211 = SACrandom( SAC_ND_ARG_in_nodesc( SAC_SET_NT_USG( FAG, (SACp_emal_6059__isaa_1146__rso_249_TheWorld, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int), SAC_ND_ARG_in_nodesc( SAC_SET_NT_USG( FAG, (SACp_emal_6058__flat_215, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int));

    SAC_ND_ALLOC__DESC((SACp_flat_211, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
    /*
     * ND_REFRESH__MIRROR( (SACp_flat_211, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_NOOP()

    SAC_ND_SET__RC((SACp_flat_211, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1)
    SAC_ND_ALLOC_BEGIN((SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 0, float)
    /*
     * ND_SET__SHAPE_arr( (SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 0), 306, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 0, float)
    SAC_ND_PRF_S__DATA((SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), SAC_ND_PRF_TOF, SAC_ND_READ((SACp_flat_211, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
    SAC_ND_DEC_RC_FREE((SACp_flat_211, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, )
    SAC_ND_PRF_SxS__DATA((SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), SAC_ND_PRF_DIV, SAC_ND_READ((SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0), 2147483648.0f)
    SAC_CUDA_ALLOC_BEGIN((SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 1, 3, float)
    /*
     * ND_SET__SHAPE_arr( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 3, 500, 26, 72)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))) == 3), 222, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 0) == 500), 222, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 1) == 26), 222, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 2) == 72), 222, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_CUDA_ALLOC_END((SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 1, 3, float)
    SAC_ND_ALLOC_BEGIN((SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 1, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 3)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 1), 222, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) == 3), 222, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 1, int)
    SAC_ND_ALLOC_BEGIN((SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 222, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    SAC_ND_ALLOC_BEGIN((SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 222, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    SAC_ND_ALLOC_BEGIN((SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 222, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    SAC_ND_ALLOC_BEGIN((SACp_wlidx_5687_X, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_wlidx_5687_X, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_wlidx_5687_X, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 222, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_wlidx_5687_X, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * CUDA_GRID_BLOCK( 9, SAC_ND_READ( (SACp_emal_6052__cnstass_5977_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ( (SACp_emal_6051__cnstass_5976_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ( (SACp_emal_6050__cnstass_5975_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ( (SACp_emal_6055__cnstass_5980_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ( (SACp_emal_6054__cnstass_5979_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ( (SACp_emal_6053__cnstass_5978_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 0, 0, 0)
     */
    {
        dim3 grid((SAC_ND_READ( (SACp_emal_6051__cnstass_5976_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)-SAC_ND_READ( (SACp_emal_6054__cnstass_5979_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)), (SAC_ND_READ( (SACp_emal_6050__cnstass_5975_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)-SAC_ND_READ( (SACp_emal_6053__cnstass_5978_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)));
        if (grid.x > 2147483647 || grid.y > 65535 || grid.z > 65535) {
            SAC_RuntimeError("CUDA XYZ grid dimension exceeds compute compatibilities max value: 2147483647 x 65535 x 65535");
        }
        dim3 block((SAC_ND_READ( (SACp_emal_6052__cnstass_5977_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)-SAC_ND_READ( (SACp_emal_6055__cnstass_5980_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)));        if (block.x > 2147483647 || block.y > 65535 || block.z > 65535) {
            SAC_RuntimeError("CUDA XYZ block dimension exceeds compute compatibilities max value: 2147483647 x 65535 x 65535");
        }
        
    /*
     * CUDA_GLOBALFUN_AP( SACf__MAIN___cuknl_6984_CUDA__i__i__i__i__i__i__fd_500_26_72__f, 8, in, float, 0, SAC_SET_NT_USG( FAG, (SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))), inout, float, 3, SAC_SET_NT_USG( FAG, (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_6052__cnstass_5977_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_6051__cnstass_5976_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_6050__cnstass_5975_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_6055__cnstass_5980_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_6054__cnstass_5979_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_6053__cnstass_5978_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))
     */
        SACf__MAIN___cuknl_6984_CUDA__i__i__i__i__i__i__fd_500_26_72__f<<<grid, block>>>(    SAC_CUDA_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))), float),     SAC_CUDA_ARG_inout( SAC_SET_NT_USG( FAG, (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), float), SAC_ND_A_MIRROR_SHAPE(SAC_SET_NT_USG( FAG, (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), 0), SAC_ND_A_MIRROR_SHAPE(SAC_SET_NT_USG( FAG, (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), 1), SAC_ND_A_MIRROR_SHAPE(SAC_SET_NT_USG( FAG, (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), 2), SAC_ND_A_MIRROR_SIZE(SAC_SET_NT_USG( FAG, (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))))), SAC_ND_A_MIRROR_DIM(SAC_SET_NT_USG( FAG, (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))))),     SAC_CUDA_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_6052__cnstass_5977_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int),     SAC_CUDA_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_6051__cnstass_5976_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int),     SAC_CUDA_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_6050__cnstass_5975_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int),     SAC_CUDA_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_6055__cnstass_5980_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int),     SAC_CUDA_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_6054__cnstass_5979_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int),     SAC_CUDA_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_6053__cnstass_5978_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int));
SAC_CUDA_GET_LAST_KERNEL_ERROR();
    }

    /*
     * ND_REFRESH__MIRROR( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 3)
     */
    SAC_NOOP()

    SAC_ND_FREE((SACp_emal_6052__cnstass_5977_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6051__cnstass_5976_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6050__cnstass_5975_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6055__cnstass_5980_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6054__cnstass_5979_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6053__cnstass_5978_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    SAC_ND_FREE((SACp_wlidx_5687_X, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_ALLOC_BEGIN((SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 3, float)
    /*
     * ND_SET__SHAPE_arr( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 3, 500, 26, 72)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 3), 222, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) == 500), 222, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1) == 26), 222, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2) == 72), 222, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 3, float)
    /*
     * CUDA_MEM_TRANSFER( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), float, hipMemcpyDeviceToHost)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SIZE( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == SAC_ND_A_SIZE( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))))), 222, "hipMemcpy: Destionation and source arrays should have equal sizes!");
    SAC_CUDA_MEM_TRANSFER((SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), float, hipMemcpyDeviceToHost)
    SAC_CUDA_FREE((SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), )
    /*
     * ND_FUN_AP( SACrandom, SACp_flat_222, 2, in_nodesc, int, SAC_SET_NT_USG( FAG, (SACp_emal_6059__isaa_1146__rso_249_TheWorld, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in_nodesc, int, SAC_SET_NT_USG( FAG, (SACp_emal_6058__flat_215, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))
     */
    SACp_flat_222 = SACrandom( SAC_ND_ARG_in_nodesc( SAC_SET_NT_USG( FAG, (SACp_emal_6059__isaa_1146__rso_249_TheWorld, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int), SAC_ND_ARG_in_nodesc( SAC_SET_NT_USG( FAG, (SACp_emal_6058__flat_215, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int));

    SAC_ND_ALLOC__DESC((SACp_flat_222, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
    /*
     * ND_REFRESH__MIRROR( (SACp_flat_222, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_NOOP()

    SAC_ND_SET__RC((SACp_flat_222, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1)
    SAC_ND_DEC_RC_FREE((SACp_emal_6058__flat_215, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, )
    SAC_ND_ALLOC_BEGIN((SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 0, float)
    /*
     * ND_SET__SHAPE_arr( (SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 0), 306, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 0, float)
    SAC_ND_PRF_S__DATA((SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), SAC_ND_PRF_TOF, SAC_ND_READ((SACp_flat_222, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0))
    SAC_ND_DEC_RC_FREE((SACp_flat_222, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, )
    SAC_ND_PRF_SxS__DATA((SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), SAC_ND_PRF_DIV, SAC_ND_READ((SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0), 2147483648.0f)
    SAC_CUDA_ALLOC_BEGIN((SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 1, 3, float)
    /*
     * ND_SET__SHAPE_arr( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 3, 500, 26, 72)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))) == 3), 222, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 0) == 500), 222, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 1) == 26), 222, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 2) == 72), 222, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_CUDA_ALLOC_END((SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 1, 3, float)
    SAC_ND_ALLOC_BEGIN((SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 1, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 3)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 1), 222, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) == 3), 222, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 1, int)
    SAC_ND_ALLOC_BEGIN((SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 222, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    SAC_ND_ALLOC_BEGIN((SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 222, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    SAC_ND_ALLOC_BEGIN((SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 222, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    SAC_ND_ALLOC_BEGIN((SACp_wlidx_5688_Y, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_wlidx_5688_Y, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_wlidx_5688_Y, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 222, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_wlidx_5688_Y, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 0, int)
    /*
     * CUDA_GRID_BLOCK( 9, SAC_ND_READ( (SACp_emal_6041__cnstass_5971_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ( (SACp_emal_6040__cnstass_5970_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ( (SACp_emal_6039__cnstass_5969_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ( (SACp_emal_6044__cnstass_5974_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ( (SACp_emal_6043__cnstass_5973_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), SAC_ND_READ( (SACp_emal_6042__cnstass_5972_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), 0, 0, 0)
     */
    {
        dim3 grid((SAC_ND_READ( (SACp_emal_6040__cnstass_5970_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)-SAC_ND_READ( (SACp_emal_6043__cnstass_5973_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)), (SAC_ND_READ( (SACp_emal_6039__cnstass_5969_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)-SAC_ND_READ( (SACp_emal_6042__cnstass_5972_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)));
        if (grid.x > 2147483647 || grid.y > 65535 || grid.z > 65535) {
            SAC_RuntimeError("CUDA XYZ grid dimension exceeds compute compatibilities max value: 2147483647 x 65535 x 65535");
        }
        dim3 block((SAC_ND_READ( (SACp_emal_6041__cnstass_5971_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)-SAC_ND_READ( (SACp_emal_6044__cnstass_5974_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0)));        if (block.x > 2147483647 || block.y > 65535 || block.z > 65535) {
            SAC_RuntimeError("CUDA XYZ block dimension exceeds compute compatibilities max value: 2147483647 x 65535 x 65535");
        }
        
    /*
     * CUDA_GLOBALFUN_AP( SACf__MAIN___cuknl_6985_CUDA__i__i__i__i__i__i__fd_500_26_72__f, 8, in, float, 0, SAC_SET_NT_USG( FAG, (SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))), inout, float, 3, SAC_SET_NT_USG( FAG, (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_6041__cnstass_5971_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_6040__cnstass_5970_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_6039__cnstass_5969_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_6044__cnstass_5974_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_6043__cnstass_5973_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, 0, SAC_SET_NT_USG( FAG, (SACp_emal_6042__cnstass_5972_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))
     */
        SACf__MAIN___cuknl_6985_CUDA__i__i__i__i__i__i__fd_500_26_72__f<<<grid, block>>>(    SAC_CUDA_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))), float),     SAC_CUDA_ARG_inout( SAC_SET_NT_USG( FAG, (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), float), SAC_ND_A_MIRROR_SHAPE(SAC_SET_NT_USG( FAG, (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), 0), SAC_ND_A_MIRROR_SHAPE(SAC_SET_NT_USG( FAG, (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), 1), SAC_ND_A_MIRROR_SHAPE(SAC_SET_NT_USG( FAG, (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), 2), SAC_ND_A_MIRROR_SIZE(SAC_SET_NT_USG( FAG, (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))))), SAC_ND_A_MIRROR_DIM(SAC_SET_NT_USG( FAG, (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))))),     SAC_CUDA_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_6041__cnstass_5971_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int),     SAC_CUDA_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_6040__cnstass_5970_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int),     SAC_CUDA_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_6039__cnstass_5969_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int),     SAC_CUDA_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_6044__cnstass_5974_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int),     SAC_CUDA_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_6043__cnstass_5973_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int),     SAC_CUDA_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_6042__cnstass_5972_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int));
SAC_CUDA_GET_LAST_KERNEL_ERROR();
    }

    /*
     * ND_REFRESH__MIRROR( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 3)
     */
    SAC_NOOP()

    SAC_ND_FREE((SACp_emal_6041__cnstass_5971_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6040__cnstass_5970_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6039__cnstass_5969_ub, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6044__cnstass_5974_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6043__cnstass_5973_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6042__cnstass_5972_lb, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_FREE((SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    SAC_ND_FREE((SACp_wlidx_5688_Y, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_ALLOC_BEGIN((SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 3, float)
    /*
     * ND_SET__SHAPE_arr( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 3, 500, 26, 72)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 3), 222, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) == 500), 222, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1) == 26), 222, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2) == 72), 222, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1, 3, float)
    /*
     * CUDA_MEM_TRANSFER( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), float, hipMemcpyDeviceToHost)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SIZE( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == SAC_ND_A_SIZE( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))))), 222, "hipMemcpy: Destionation and source arrays should have equal sizes!");
    SAC_CUDA_MEM_TRANSFER((SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), float, hipMemcpyDeviceToHost)
    SAC_CUDA_FREE((SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), )
    SAC_ND_ALLOC_BEGIN((SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), 1, 1, unsigned char)
    /*
     * ND_SET__SHAPE_arr( (SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), 1, 5)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, ))))))))))) == 1), 1, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), 0) == 5), 1, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), 1, 1, unsigned char)
    SAC_ND_CREATE__STRING__DATA((SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), "work")
    /*
     * ND_ASSIGN( (SACp_tcp_6988__emal_6033__flat_228, (AKD, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), -3, (SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), 1, )
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_tcp_6988__emal_6033__flat_228, (AKD, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, ))))))))))) == 1), 38, "Assignment with incompatible types found!");
    SAC_ND_A_DESC( (SACp_tcp_6988__emal_6033__flat_228, (AKD, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, ))))))))))) = SAC_ND_A_DESC( (SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))));
    SAC_ND_A_MIRROR_SHAPE( (SACp_tcp_6988__emal_6033__flat_228, (AKD, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), 0) = SAC_ND_A_SHAPE( (SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), 0);
    SAC_ND_A_MIRROR_SIZE( (SACp_tcp_6988__emal_6033__flat_228, (AKD, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, ))))))))))) = SAC_ND_A_SIZE( (SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))));
    SAC_ND_A_DESC_SHAPE( (SACp_tcp_6988__emal_6033__flat_228, (AKD, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), 0) = SAC_ND_A_SHAPE( (SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), 0);
    SAC_ND_A_DESC_SIZE( (SACp_tcp_6988__emal_6033__flat_228, (AKD, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, ))))))))))) = SAC_ND_A_SIZE( (SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))));
    SAC_ND_ASSIGN__DATA( (SACp_tcp_6988__emal_6033__flat_228, (AKD, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), (SACp_emal_6033__flat_228, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, )))))))))), )

    /*
     * ND_FUN_AP( to_string, , 3, out, SACt_String__string, SAC_SET_NT_USG( FAG, (SACp_flat_227, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), in, unsigned char, SAC_SET_NT_USG( FAG, (SACp_tcp_6988__emal_6033__flat_228, (AKD, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, ))))))))))), in_nodesc, int, SAC_SET_NT_USG( FAG, (SACp_emal_6034__flat_234, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))
     */
    SAC_ND_FUNAP2( to_string,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACp_flat_227, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), SACt_String__string), SAC_ND_ARG_in( SAC_SET_NT_USG( FAG, (SACp_tcp_6988__emal_6033__flat_228, (AKD, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (UCH, ))))))))))), unsigned char), SAC_ND_ARG_in_nodesc( SAC_SET_NT_USG( FAG, (SACp_emal_6034__flat_234, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int))

    /*
     * ND_REFRESH__MIRROR( (SACp_flat_227, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 0)
     */
    SAC_NOOP()

    SAC_ND_DEC_RC_FREE((SACp_emal_6034__flat_234, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, )
    /*
     * ND_FUN_AP( SACf_C99Benchmarking__getInterval__SACt_String__string__i__i, , 4, out, SACt_Interval__Interval, SAC_SET_NT_USG( FAG, (SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), in, SACt_String__string, SAC_SET_NT_USG( FAG, (SACp_flat_227, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), in, int, SAC_SET_NT_USG( FAG, (SACp_emal_6059__isaa_1146__rso_249_TheWorld, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), in, int, SAC_SET_NT_USG( FAG, (SACp_emal_6035__flat_236, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_C99Benchmarking__getInterval__SACt_String__string__i__i,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), SACt_Interval__Interval), SAC_ND_ARG_in( SAC_SET_NT_USG( FAG, (SACp_flat_227, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), SACt_String__string), SAC_ND_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_6059__isaa_1146__rso_249_TheWorld, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int), SAC_ND_ARG_in( SAC_SET_NT_USG( FAG, (SACp_emal_6035__flat_236, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))), int))

    /*
     * ND_REFRESH__MIRROR( (SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 0)
     */
    SAC_NOOP()

    /*
     * ND_FUN_AP( SACf_C99Benchmarking__start__SACt_C99Benchmarking__Interval, , 1, inout, SACt_Interval__Interval, SAC_SET_NT_USG( FAG, (SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_C99Benchmarking__start__SACt_C99Benchmarking__Interval,  SAC_ND_ARG_inout( SAC_SET_NT_USG( FAG, (SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), SACt_Interval__Interval))

    /*
     * ND_ASSIGN( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -5, (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 3, )
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 3), 41, "Assignment with incompatible types found!");
    SAC_ND_A_DESC( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = SAC_ND_A_DESC( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))));
    SAC_ND_A_MIRROR_SHAPE( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = SAC_ND_A_SHAPE( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);
    SAC_ND_A_MIRROR_SHAPE( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1) = SAC_ND_A_SHAPE( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1);
    SAC_ND_A_MIRROR_SHAPE( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2) = SAC_ND_A_SHAPE( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2);
    SAC_ND_A_MIRROR_SIZE( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = SAC_ND_A_SIZE( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))));
    SAC_ND_A_DESC_SHAPE( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = SAC_ND_A_SHAPE( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);
    SAC_ND_A_DESC_SHAPE( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1) = SAC_ND_A_SHAPE( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1);
    SAC_ND_A_DESC_SHAPE( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2) = SAC_ND_A_SHAPE( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2);
    SAC_ND_A_DESC_SIZE( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = SAC_ND_A_SIZE( (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))));
    SAC_ND_ASSIGN__DATA( (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), (SACp_emal_6036_Y, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )

    /*
     * ND_ASSIGN( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -5, (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 3, )
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) == 3), 41, "Assignment with incompatible types found!");
    SAC_ND_A_DESC( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = SAC_ND_A_DESC( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))));
    SAC_ND_A_MIRROR_SHAPE( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = SAC_ND_A_SHAPE( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);
    SAC_ND_A_MIRROR_SHAPE( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1) = SAC_ND_A_SHAPE( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1);
    SAC_ND_A_MIRROR_SHAPE( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2) = SAC_ND_A_SHAPE( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2);
    SAC_ND_A_MIRROR_SIZE( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = SAC_ND_A_SIZE( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))));
    SAC_ND_A_DESC_SHAPE( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = SAC_ND_A_SHAPE( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);
    SAC_ND_A_DESC_SHAPE( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1) = SAC_ND_A_SHAPE( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1);
    SAC_ND_A_DESC_SHAPE( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2) = SAC_ND_A_SHAPE( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2);
    SAC_ND_A_DESC_SIZE( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = SAC_ND_A_SIZE( (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))));
    SAC_ND_ASSIGN__DATA( (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), (SACp_emal_6047_X, (AKS, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )

    /*
     * ND_FUN_AP( SACf__MAIN__tbmm__f_X_X_X__f_X_X_X, , 3, out, float, SAC_SET_NT_USG( FAG, (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))), in, float, SAC_SET_NT_USG( FAG, (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))), in, float, SAC_SET_NT_USG( FAG, (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf__MAIN__tbmm__f_X_X_X__f_X_X_X,  SAC_ND_ARG_out( SAC_SET_NT_USG( FAG, (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))), float), SAC_ND_ARG_in( SAC_SET_NT_USG( FAG, (SACp_tcp_6986__emal_6047_X, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))), float), SAC_ND_ARG_in( SAC_SET_NT_USG( FAG, (SACp_tcp_6987__emal_6036_Y, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))), float))

    /*
     * ND_REFRESH__MIRROR( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -5)
     */
    SAC_ND_A_MIRROR_SHAPE( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0) = SAC_ND_A_DESC_SHAPE( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0);
    SAC_ND_A_MIRROR_SHAPE( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1) = SAC_ND_A_DESC_SHAPE( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1);
    SAC_ND_A_MIRROR_SHAPE( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2) = SAC_ND_A_DESC_SHAPE( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2);
    SAC_ND_A_MIRROR_SIZE( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))));

    SAC_ND_ALLOC_BEGIN((SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 1, int)
    /*
     * ND_SET__SHAPE_arr( (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 3)
     */
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 1), 41, "Assignment with incompatible types found!");
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_SHAPE( (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) == 3), 41, "Assignment with incompatible types found!");
    SAC_NOOP()

    SAC_ND_ALLOC_END((SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, 1, int)
    /*
     * ND_PRF_SHAPE_A__DATA( (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -5)
     */
    SAC_TR_PRF_PRINT( ("ND_PRF_SHAPE_A__DATA( (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -5)"))
    SAC_ND_WRITE_COPY( (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, SAC_ND_A_SHAPE( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0), );
    SAC_ND_WRITE_COPY( (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, SAC_ND_A_SHAPE( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 1), );
    SAC_ND_WRITE_COPY( (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 2, SAC_ND_A_SHAPE( (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 2), );

    /*
     * ND_FUN_AP( SACf_C99Benchmarking__end__SACt_C99Benchmarking__Interval, , 1, inout, SACt_Interval__Interval, SAC_SET_NT_USG( FAG, (SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_C99Benchmarking__end__SACt_C99Benchmarking__Interval,  SAC_ND_ARG_inout( SAC_SET_NT_USG( FAG, (SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), SACt_Interval__Interval))

    /*
     * ND_FUN_AP( SACf_C99Benchmarking__printResult__SACt_C99Benchmarking__Interval, , 1, inout, SACt_Interval__Interval, SAC_SET_NT_USG( FAG, (SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))))
     */
    SAC_ND_FUNAP2( SACf_C99Benchmarking__printResult__SACt_C99Benchmarking__Interval,  SAC_ND_ARG_inout( SAC_SET_NT_USG( FAG, (SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, ))))))))))), SACt_Interval__Interval))

    SAC_ND_DEC_RC_FREE((SACl_interval, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (OTH, )))))))))), 1, )
    /*
     * ND_IDXS2OFFSET_id( (SACp_emal_6031__ivesli_5759, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACp_emal_6059__isaa_1146__rso_249_TheWorld, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_emal_6059__isaa_1146__rso_249_TheWorld, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_emal_6059__isaa_1146__rso_249_TheWorld, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
     */
    SAC_ND_WRITE( (SACp_emal_6031__ivesli_5759, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = ( SAC_ND_READ( (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 2) * ( SAC_ND_READ( (SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1) * SAC_ND_READ( (SACp_emal_6059__isaa_1146__rso_249_TheWorld, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) + SAC_ND_READ( (SACp_emal_6059__isaa_1146__rso_249_TheWorld, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) ) + SAC_ND_READ( (SACp_emal_6059__isaa_1146__rso_249_TheWorld, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) );

    SAC_ND_DEC_RC_FREE((SACp_emal_6059__isaa_1146__rso_249_TheWorld, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1, )
    SAC_ND_FREE((SACp_emal_6032__isaa_1213_Z, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    /*
     * ND_PRF_IDX_SEL__DATA( (SACp_emal_6030__pinl_698__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -5, (SACp_emal_6031__ivesli_5759, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
     */
    SAC_TR_PRF_PRINT( ("ND_PRF_IDX_SEL__DATA( (SACp_emal_6030__pinl_698__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), -5, (SACp_emal_6031__ivesli_5759, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))"))
    SAC_ASSURE_TYPE_LINE ((SAC_ND_A_DIM( (SACp_emal_6031__ivesli_5759, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) == 0), 152, "1st argument of _idx_sel_ is not a scalar!");
    SAC_ND_WRITE_READ_COPY( (SACp_emal_6030__pinl_698__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0, (SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), SAC_ND_READ( (SACp_emal_6031__ivesli_5759, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), )

    SAC_ND_FREE((SACl_Z, (AKD, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    SAC_ND_FREE((SACp_emal_6031__ivesli_5759, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_ND_PRF_S__DATA((SACp_emal_6029__pinl_701__flat_252, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), SAC_ND_PRF_TOI, SAC_ND_READ((SACp_emal_6030__pinl_698__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), 0))
    SAC_ND_FREE((SACp_emal_6030__pinl_698__flat_140, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), )
    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_emal_6029__pinl_701__flat_252, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), (SACp_emal_6029__pinl_701__flat_252, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACf__MAIN__main, , 1, out, int, (SAC_arg_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * WITH-loop Count: 0
 * _MAIN:_INIT::SACf__MAIN_CL_INIT__init(...) [ body ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACf__MAIN_CL_INIT__init, , 0)
 */
SAC_ND_DEF_FUN_BEGIN2( SACf__MAIN_CL_INIT__init, void, void)
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    SAC_INIT_LOCAL_MEM()
    /*
     * ND_FUN_AP( SACf_World_CL_INIT__init_TheWorld__SACt_World__World, , 1, inout, SACt_World__World, SAC_SET_NT_USG( TFA, (SACo_World__TheWorld, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, ))))))))))
     */
    SAC_ND_FUNAP2( SACf_World_CL_INIT__init_TheWorld__SACt_World__World,  SAC_ND_ARG_inout( SAC_SET_NT_USG( TFA, (SACo_World__TheWorld, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, ))))))))), SACt_World__World))

    /*
     * ND_FUN_AP( SACf_C99Benchmarking_CL_INIT__init_TheBenchmarkObject__SACt_C99Benchmarking__C99Benchmarking, , 1, inout, SACt_C99Benchmarking__C99Benchmarking, SAC_SET_NT_USG( TFA, (SACo_C99Benchmarking__TheBenchmarkObject, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, ))))))))))
     */
    SAC_ND_FUNAP2( SACf_C99Benchmarking_CL_INIT__init_TheBenchmarkObject__SACt_C99Benchmarking__C99Benchmarking,  SAC_ND_ARG_inout( SAC_SET_NT_USG( TFA, (SACo_C99Benchmarking__TheBenchmarkObject, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, ))))))))), SACt_C99Benchmarking__C99Benchmarking))

    /*
     * ND_FUN_AP( SACf_MTClock_CL_INIT__init_TheMTClock__SACt_MTClock__MTClock, , 1, inout, SACt_MTClock__MTClock, SAC_SET_NT_USG( TFA, (SACo_MTClock__TheMTClock, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, ))))))))))
     */
    SAC_ND_FUNAP2( SACf_MTClock_CL_INIT__init_TheMTClock__SACt_MTClock__MTClock,  SAC_ND_ARG_inout( SAC_SET_NT_USG( TFA, (SACo_MTClock__TheMTClock, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, ))))))))), SACt_MTClock__MTClock))

    /*
     * ND_FUN_AP( SACf_Terminal_CL_INIT__init_TheTerminal__SACt_Terminal__Terminal, , 1, inout, SACt_Terminal__Terminal, SAC_SET_NT_USG( TFA, (SACo_Terminal__TheTerminal, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, ))))))))))
     */
    SAC_ND_FUNAP2( SACf_Terminal_CL_INIT__init_TheTerminal__SACt_Terminal__Terminal,  SAC_ND_ARG_inout( SAC_SET_NT_USG( TFA, (SACo_Terminal__TheTerminal, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, ))))))))), SACt_Terminal__Terminal))

    /*
     * ND_FUN_AP( SACf_TermFile_CL_INIT__init_stdout__SACt_TermFile__TermFile, , 1, inout, SACt_TermFile__TermFile, SAC_SET_NT_USG( TFA, (SACo_TermFile__stdout, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, ))))))))))
     */
    SAC_ND_FUNAP2( SACf_TermFile_CL_INIT__init_stdout__SACt_TermFile__TermFile,  SAC_ND_ARG_inout( SAC_SET_NT_USG( TFA, (SACo_TermFile__stdout, (SCL, (HID, (NUQ, (INT, (GLO, (NON, (NOT, ))))))))), SACt_TermFile__TermFile))

    /*
     * ND_FUN_RET( , 0)
     */
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACf__MAIN_CL_INIT__init, , 0)
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * WITH-loop Count: 0
 * _MAIN:sacprelude_p::SACf__MAIN_CLsacprelude_p__zero__f_X_X_X(...) [ body ]
 ****************************************************************************/
/*
 * ND_FUN_DEF_BEGIN( SACf__MAIN_CLsacprelude_p__zero__f_X_X_X, , 2, out, float, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_A, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
 */
SAC_ND_DEF_FUN_BEGIN2( SACf__MAIN_CLsacprelude_p__zero__f_X_X_X, void,  SAC_ND_PARAM_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_ND_PARAM_in( (SACl_A, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float))
{
  SAC_HM_DEFINE_THREAD_STATUS( SAC_HM_single_threaded)
  SAC_MT_DEFINE_ST_SELF()

  { 
    SAC_ND_DECL_CONST__DATA((SACp_emal_6940__flat_284, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))), float, 0.0f)

    /*
     * ND_DECL__MIRROR_PARAM( (SACl_A, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), -5)
     */
    int SAC_ND_A_MIRROR_SHAPE( (SACl_A, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0) = SAC_ND_A_DESC_SHAPE( (SACl_A, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0);
    int SAC_ND_A_MIRROR_SHAPE( (SACl_A, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 1) = SAC_ND_A_DESC_SHAPE( (SACl_A, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 1);
    int SAC_ND_A_MIRROR_SHAPE( (SACl_A, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 2) = SAC_ND_A_DESC_SHAPE( (SACl_A, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 2);
    int SAC_ND_A_MIRROR_SIZE( (SACl_A, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, ))))))))))) = SAC_ND_A_DESC_SIZE( (SACl_A, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))));
    const int SAC_ND_A_MIRROR_DIM( (SACl_A, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, ))))))))))) = 3;

    SAC_INIT_LOCAL_MEM()
    SAC_ND_DEC_RC_FREE((SACl_A, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 1, )
    /*
     * ND_FUN_RET( , 1, out, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), (SACp_emal_6940__flat_284, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))))
     */
    SAC_ND_RET_out( (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), (SACp_emal_6940__flat_284, (SCL, (NHD, (NUQ, (FLO, (GLO, (NON, (NOT, (NDI, (FLO, )))))))))))
    return;
    SAC_CLEANUP_LOCAL_MEM()
  }
/*
   * ND_FUN_DEF_END( SACf__MAIN_CLsacprelude_p__zero__f_X_X_X, , 2, out, float, (SAC_arg_1, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), in, float, (SACl_A, (AKD, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))))
   */
}
SAC_ND_FUN_DEF_END2()



/****************************************************************************
 * WITH-loop Count: 0
 * _MAIN::SACf__MAIN___cuknl_6985_CUDA__i__i__i__i__i__i__fd_500_26_72__f(...) [ body ]
 ****************************************************************************/
/*
 * CUDA_GLOBALFUN_DEF_BEGIN( SACf__MAIN___cuknl_6985_CUDA__i__i__i__i__i__i__fd_500_26_72__f, 8, in, float, (SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0, inout, float, (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 3, in, int, (SACp_ub_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0)
 */
__global__ void SACf__MAIN___cuknl_6985_CUDA__i__i__i__i__i__i__fd_500_26_72__f(SAC_CUDA_PARAM_in( (SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_CUDA_PARAM_inout( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), float), int SAC_ND_A_MIRROR_SHAPE((SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 0), int SAC_ND_A_MIRROR_SHAPE((SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 1), int SAC_ND_A_MIRROR_SHAPE((SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 2), int SAC_ND_A_MIRROR_SIZE((SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, ))))))))))), int SAC_ND_A_MIRROR_DIM((SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, ))))))))))), SAC_CUDA_PARAM_in( (SACp_ub_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_ub_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_ub_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_lb_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_lb_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_lb_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int)){

{ 
  /*
   * ND_DECL( (SACp_wlidx_5688_Y, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
   */
  SAC_ND_DECL__DATA( (SACp_wlidx_5688_Y, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
  SAC_ND_DECL__DESC( (SACp_wlidx_5688_Y, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_NOTHING()

  /*
   * ND_DECL( (SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
   */
  SAC_ND_DECL__DATA( (SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
  SAC_ND_DECL__DESC( (SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_NOTHING()

  /*
   * ND_DECL( (SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
   */
  SAC_ND_DECL__DATA( (SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
  SAC_ND_DECL__DESC( (SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_NOTHING()

  /*
   * ND_DECL( (SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
   */
  SAC_ND_DECL__DATA( (SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
  SAC_ND_DECL__DESC( (SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_NOTHING()

  /*
   * CUDA_DECL_KERNEL_ARRAY( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 1, 3)
   */
  int SAC_ND_A_FIELD( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))[3];
  SAC_ND_DECL__DESC( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  const int SAC_ND_A_MIRROR_SHAPE( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 3;
  const int SAC_ND_A_MIRROR_SIZE( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 3;
  const int SAC_ND_A_MIRROR_DIM( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;


  SAC_INIT_LOCAL_MEM()
  /*
   * CUDA_WLIDS( (SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 3, 2, (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), false)
   */
  SAC_CUDA_WLIDS_HD( (SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, THREADIDX_X, SACp_step_2, SACp_width_2, SACp_lb_2, SACp_ub_2)
SAC_ND_WRITE( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 2) = SAC_ND_READ( (SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

  /*
   * CUDA_WLIDS( (SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 3, 1, (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), false)
   */
  SAC_CUDA_WLIDS_HD( (SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, BLOCKIDX_X, SACp_step_1, SACp_width_1, SACp_lb_1, SACp_ub_1)
SAC_ND_WRITE( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1) = SAC_ND_READ( (SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

  /*
   * CUDA_WLIDS( (SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 3, 0, (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), false)
   */
  SAC_CUDA_WLIDS_HD( (SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, BLOCKIDX_Y, SACp_step_0, SACp_width_0, SACp_lb_0, SACp_ub_0)
SAC_ND_WRITE( (SACp_pinl_670_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = SAC_ND_READ( (SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

  /*
   * ND_IDXS2OFFSET_arr( (SACp_wlidx_5688_Y, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, 500, 26, 72)
   */
  SAC_ND_WRITE( (SACp_wlidx_5688_Y, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = ( 72 * ( 26 * SAC_ND_READ( (SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) + SAC_ND_READ( (SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) ) + SAC_ND_READ( (SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) );

  /*
   * CUDA_WL_ASSIGN( (SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 3, (SACp_wlidx_5688_Y, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
   */
  SAC_ND_WRITE_READ_COPY( (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), SAC_ND_READ( (SACp_wlidx_5688_Y, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), (SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0, );

  SAC_ND_FREE((SACp_wlidx_5688_Y, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_ND_FREE((SACp_pinl_671__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_ND_FREE((SACp_pinl_672__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_ND_FREE((SACp_pinl_673__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOOP()

  SAC_CLEANUP_LOCAL_MEM()
}
/*
 * CUDA_GLOBALFUN_DEF_END( SACf__MAIN___cuknl_6985_CUDA__i__i__i__i__i__i__fd_500_26_72__f, 8, in, float, (SACp_emal_6046__pinl_666__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0, inout, float, (SACp_emal_6037__iwlmem_5968_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 3, in, int, (SACp_ub_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0)
 */
}



/****************************************************************************
 * WITH-loop Count: 0
 * _MAIN::SACf__MAIN___cuknl_6984_CUDA__i__i__i__i__i__i__fd_500_26_72__f(...) [ body ]
 ****************************************************************************/
/*
 * CUDA_GLOBALFUN_DEF_BEGIN( SACf__MAIN___cuknl_6984_CUDA__i__i__i__i__i__i__fd_500_26_72__f, 8, in, float, (SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0, inout, float, (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 3, in, int, (SACp_ub_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0)
 */
__global__ void SACf__MAIN___cuknl_6984_CUDA__i__i__i__i__i__i__fd_500_26_72__f(SAC_CUDA_PARAM_in( (SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), float), SAC_CUDA_PARAM_inout( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), float), int SAC_ND_A_MIRROR_SHAPE((SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 0), int SAC_ND_A_MIRROR_SHAPE((SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 1), int SAC_ND_A_MIRROR_SHAPE((SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 2), int SAC_ND_A_MIRROR_SIZE((SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, ))))))))))), int SAC_ND_A_MIRROR_DIM((SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, ))))))))))), SAC_CUDA_PARAM_in( (SACp_ub_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_ub_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_ub_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_lb_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_lb_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int), SAC_CUDA_PARAM_in( (SACp_lb_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), int)){

{ 
  /*
   * ND_DECL( (SACp_wlidx_5687_X, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
   */
  SAC_ND_DECL__DATA( (SACp_wlidx_5687_X, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
  SAC_ND_DECL__DESC( (SACp_wlidx_5687_X, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_NOTHING()

  /*
   * ND_DECL( (SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
   */
  SAC_ND_DECL__DATA( (SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
  SAC_ND_DECL__DESC( (SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_NOTHING()

  /*
   * ND_DECL( (SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
   */
  SAC_ND_DECL__DATA( (SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
  SAC_ND_DECL__DESC( (SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_NOTHING()

  /*
   * ND_DECL( (SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 0)
   */
  SAC_ND_DECL__DATA( (SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
  SAC_ND_DECL__DESC( (SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_NOTHING()

  /*
   * CUDA_DECL_KERNEL_ARRAY( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, 1, 3)
   */
  int SAC_ND_A_FIELD( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))[3];
  SAC_ND_DECL__DESC( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  const int SAC_ND_A_MIRROR_SHAPE( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = 3;
  const int SAC_ND_A_MIRROR_SIZE( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 3;
  const int SAC_ND_A_MIRROR_DIM( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, ))))))))))) = 1;


  SAC_INIT_LOCAL_MEM()
  /*
   * CUDA_WLIDS( (SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 3, 2, (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), false)
   */
  SAC_CUDA_WLIDS_HD( (SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, THREADIDX_X, SACp_step_2, SACp_width_2, SACp_lb_2, SACp_ub_2)
SAC_ND_WRITE( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 2) = SAC_ND_READ( (SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

  /*
   * CUDA_WLIDS( (SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 3, 1, (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), false)
   */
  SAC_CUDA_WLIDS_HD( (SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, BLOCKIDX_X, SACp_step_1, SACp_width_1, SACp_lb_1, SACp_ub_1)
SAC_ND_WRITE( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 1) = SAC_ND_READ( (SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

  /*
   * CUDA_WLIDS( (SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, 3, 0, (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), false)
   */
  SAC_CUDA_WLIDS_HD( (SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0, BLOCKIDX_Y, SACp_step_0, SACp_width_0, SACp_lb_0, SACp_ub_0)
SAC_ND_WRITE( (SACp_pinl_648_iv, (AKS, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = SAC_ND_READ( (SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0);

  /*
   * ND_IDXS2OFFSET_arr( (SACp_wlidx_5687_X, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, (SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), (SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 3, 500, 26, 72)
   */
  SAC_ND_WRITE( (SACp_wlidx_5687_X, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) = ( 72 * ( 26 * SAC_ND_READ( (SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) + SAC_ND_READ( (SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) ) + SAC_ND_READ( (SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0) );

  /*
   * CUDA_WL_ASSIGN( (SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0, (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 3, (SACp_wlidx_5687_X, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))))
   */
  SAC_ND_WRITE_READ_COPY( (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), SAC_ND_READ( (SACp_wlidx_5687_X, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0), (SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0, );

  SAC_ND_FREE((SACp_wlidx_5687_X, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_ND_FREE((SACp_pinl_649__eat_265, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_ND_FREE((SACp_pinl_650__eat_266, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_ND_FREE((SACp_pinl_651__eat_267, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
    SAC_NOOP()

  SAC_CLEANUP_LOCAL_MEM()
}
/*
 * CUDA_GLOBALFUN_DEF_END( SACf__MAIN___cuknl_6984_CUDA__i__i__i__i__i__i__fd_500_26_72__f, 8, in, float, (SACp_emal_6057__pinl_644__flat_349, (SCL, (NHD, (NUQ, (FLO, (GLO, (FPM, (NOT, (NDI, (FLO, )))))))))), 0, inout, float, (SACp_emal_6048__iwlmem_5967_dev, (AKS, (NHD, (NUQ, (FLO, (GLO, (FPO, (NOT, (NDI, (OTH, )))))))))), 3, in, int, (SACp_ub_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_ub_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_2, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_1, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0, in, int, (SACp_lb_0, (SCL, (NHD, (NUQ, (INT, (GLO, (FPM, (NOT, (NDI, (INT, )))))))))), 0)
 */
}

/*
 * stubs for SACARGfreeDataUdt and SACARGcopyDataUdt
 */
extern "C" void SACARGfreeDataUdt( int, void *);
extern "C" void *SACARGcopyDataUdt( int, int, void *);
void SACARGfreeDataUdt( int size, void *data) {}
void *SACARGcopyDataUdt( int type, int size, void *data) { return ((void *) 0x0); } 

int main( int __argc, char *__argv[])
{
  SAC_ND_DECL__DATA( (SAC_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int, )
  SAC_ND_DECL__DESC( (SAC_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), )
  SAC_NOTHING()
  SAC_MT_SETUP_INITIAL();
  SAC_PF_SETUP();
  SAC_HM_SETUP();
  SAC_MT_SETUP();
  SAC_CS_SETUP();
  SAC_COMMANDLINE_SET( __argc, __argv);

  SAC_INVOKE_MAIN_FUN( SACf__MAIN__main, SAC_ND_ARG_out( (SAC_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), int));

  SAC_DISTMEM_BARRIER();
  SAC_PF_PRINT();
  SAC_CS_FINALIZE();
  SAC_MT_FINALIZE();
  SAC_HM_PRINT();

  return( SAC_ND_READ( (SAC_res, (SCL, (NHD, (NUQ, (INT, (GLO, (NON, (NOT, (NDI, (INT, )))))))))), 0));
}
